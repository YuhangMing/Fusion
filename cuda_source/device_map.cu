#include "hip/hip_runtime.h"
#include "device_map.cuh"

#define UINT_MAX 0x7fffffff * 2U + 1

__device__ int DMap::Hash(const int3& pos) {

	int res = ((pos.x * 73856093) ^ (pos.y * 19349669) ^ (pos.z * 83492791));
	res %= DMap::MaxBlocks;
	if (res < 0)
		res += DMap::MaxBlocks;

	return res;
}

__device__ HVoxel* DMap::FindVoxel(const int3& pos) {
	HBlock* block = FindBlock(pos);
	if (block->ptr < 0)
		return nullptr;

	int idx = 0;
	idx += pos.z * DMap::BlockSize * DMap::BlockSize;
	idx += pos.y * DMap::BlockSize;
	idx += pos.x;
	HVoxel* voxel = &Voxels[block->ptr + idx];

	return voxel;
}

__device__ HBlock* DMap::FindBlock(const int3& pos) {

	int idx = Hash(pos);
	HBlock* block = &Blocks[idx];
	int counter = 0;

	while (block->ptr != -1) {
		if (block->pos == pos)
			return block;

		if (block->next == -1 ||
		    counter >= 5)
			return nullptr;

		counter++;
		block = &Blocks[block->next];
	}

	return nullptr;
}

__device__ int DMap::AllocateMem() {
	uint ptr = atomicAdd(StackPtr, 1);
	if (ptr >= 0 && ptr < DMap::MaxBlocks)
		return StackMem[ptr];
	else
		return -1;
}

__device__ void DMap::ReleaseMem(int idx) {
	uint ptr = atomicSub(StackPtr, 1);
	if (ptr >= 0 && ptr < DMap::MaxBlocks)
		StackMem[ptr] = idx;
}

__device__ void DMap::ResetDeviceMem(int idx) {
	if(idx >= 0 && idx < DMap::MaxVoxels) {
		if(idx == 0)
			StackPtr[0] = 0;
		Voxels[idx].w = 0;
		StackMem[idx] = idx;
	}

	if(idx >= 0 && idx < DMap::MaxBlocks) {
		Blocks[idx].ptr = -1;
		Blocks[idx].next = -1;
	}
}

__device__ ORBKey* KeyMap::FindKey(const float3& pos) {

	float3 gridPos = pos / GridSize;
	int idx = Hash(make_int3(gridPos.x, gridPos.y, gridPos.z));
	int bucketIdx = idx * nBuckets;
	for(int i = 0; i < nBuckets; ++i, ++bucketIdx) {
		ORBKey* key = &Keys[bucketIdx];
		if(norm(key->pos - pos) <= 10) {
			return key;
		}
	}
	return nullptr;
}

__device__ void KeyMap::InsertKey(ORBKey* key) {

	ORBKey* oldKey = FindKey(key->pos);
	if(!oldKey) {

	}
	else {

	}
}
