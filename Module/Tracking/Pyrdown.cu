#include "hip/hip_runtime.h"
#include "cufunc.h"
#include "Reduction.h"

__constant__ float sigSpace = 0.5 / (4 * 4);
__constant__ float sigRange = 0.5 / (0.5 * 0.5);
__global__ void FilterDepthKernel(const PtrStepSz<unsigned short> depth,
		PtrStep<float> filteredDepth, float depthScaleInv) {

	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= depth.cols || y >= depth.rows)
		return;

    float center = depth.ptr(y)[x] * depthScaleInv;
	if(isnan(center)) {
		filteredDepth.ptr(y)[x] = center;
		return;
	}

    int R = 2;
    int D = R * 2 + 1;
    int tx = min (x - D/2 + D, depth.cols - 1);
    int ty = min (y - D/2 + D, depth.rows - 1);

    float sum1 = 0;
    float sum2 = 0;
    for (int cy = max(y - D / 2, 0); cy < ty; ++cy) {
		for (int cx = max(x - D / 2, 0); cx < tx; ++cx) {
			float val = depth.ptr(cy)[cx] * depthScaleInv;
			float space2 = (x - cx) * (x - cx) + (y - cy) * (y - cy);
			float color2 = (center - val) * (center - val);
			float weight = exp(-(space2 * sigSpace + color2 * sigRange));
			sum1 += val * weight;
			sum2 += weight;
		}
    }

    filteredDepth.ptr(y)[x] = sum1 / sum2;
}

void FilterDepth(const DeviceArray2D<unsigned short> & depth,
		DeviceArray2D<float> & filteredDepth, float depthScale) {

	dim3 thread(8, 8);
	dim3 block(DivUp(depth.cols, thread.x), DivUp(depth.rows, thread.y));

	FilterDepthKernel<<<block, thread>>>(depth, filteredDepth, 1.0 / depthScale);
}

__global__ void ComputeVMapKernel(const PtrStepSz<float> depth,
		PtrStep<float4> vmap, float invfx, float invfy, float cx, float cy,
		float depthCutoff) {

	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= depth.cols || y >= depth.rows)
		return;

	float4 v;
	v.z = depth.ptr(y)[x];
	if(!isnan(v.z) && v.z > 0.1 && v.z < depthCutoff) {
		v.x = v.z * (x - cx) * invfx;
		v.y = v.z * (y - cy) * invfy;
		v.w = 1.0;
	}
	else
		v.x = __int_as_float(0x7fffffff);

	vmap.ptr(y)[x] = v;
}

void ComputeVMap(const DeviceArray2D<float> & depth,
		DeviceArray2D<float4> & vmap, float fx, float fy, float cx, float cy,
		float depthCutoff) {

	dim3 thread(8, 8);
	dim3 block(DivUp(depth.cols, thread.x), DivUp(depth.rows, thread.y));

	ComputeVMapKernel<<<block, thread>>>(depth, vmap, 1.0 / fx, 1.0 / fy, cx, cy, depthCutoff);
}

__global__ void ComputeNMapKernel(const PtrStepSz<float4> vmap,
		PtrStepSz<float4> nmap) {

	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= vmap.cols || y >= vmap.rows)
		return;

	if (x == vmap.cols - 1 || y == vmap.rows - 1) {
		nmap.ptr(y)[x] = make_float4(__int_as_float(0x7fffffff));
		return;
	}

	float4 vcentre = vmap.ptr(y)[x];
	float4 vright = vmap.ptr(y)[x + 1];
	float4 vdown = vmap.ptr(y + 1)[x];

	if (!isnan(vcentre.x) && !isnan(vright.x) && !isnan(vdown.x)) {
		nmap.ptr(y)[x] = normalised(cross(vright - vcentre, vdown - vcentre));
	} else
		nmap.ptr(y)[x] = make_float4(__int_as_float(0x7fffffff));
}

void ComputeNMap(const DeviceArray2D<float4> & vmap, DeviceArray2D<float4> & nmap) {

	dim3 block(8, 8);
	dim3 grid(DivUp(vmap.cols, block.x), DivUp(vmap.rows, block.y));

	ComputeNMapKernel<<<grid, block>>>(vmap, nmap);
}

template<class T, class U, int size> __global__
void BilateralFiltering_device(const PtrStepSz<T> src, PtrStep<U> dst, float s,
		float r, float scale) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= src.cols || y >= src.rows)
		return;

	int minX = max(0, x - size / 2);
	int maxX = min(x + size / 2 + 1, src.cols);
	int minY = max(0, y - size / 2);
	int maxY = min(y + size / 2 + 1, src.rows);

	float val = 0, weight = 0;
	float valc = src.ptr(y)[x] * scale;
	for (int i = minX; i < maxX; ++i) {
		for (int j = minY; j < maxY; ++j) {
			float valp = src.ptr(j)[i] * scale;
			float gs2 = (x - i) * (x - i) + (y - j) * (y - j);
			float gr2 = (valc - valp) * (valc - valp);
			float wp = __expf(-gs2 * s - gr2 * r);
			val += wp * valp;
			weight += wp;
		}
	}
	if (weight < 1e-6)
		dst.ptr(y)[x] = (U) valc;
	else
		dst.ptr(y)[x] = (U) (val / weight);
}

void BilateralFiltering(const DeviceArray2D<ushort>& src,
		DeviceArray2D<float>& dst, float scale) {
	dim3 block(8, 8);
	dim3 grid(cv::divUp(src.cols, block.x), cv::divUp(src.rows, block.y));

	float SigmaSpace = 0.5 / (4 * 4);
	float SigmaRange = 0.5 / (0.5 * 0.5);
	BilateralFiltering_device<ushort, float, 5> <<<grid, block>>>(src, dst,
			SigmaSpace, SigmaRange, 1.0 / scale);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

template<class T, class U> __global__
void PyrDownGaussian_device(const PtrStepSz<T> src, PtrStepSz<U> dst,
		float* kernel) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= dst.cols || y >= dst.rows)
		return;

	const int D = 5;
	float center = src.ptr(2 * y)[2 * x];
	int tx = min(2 * x - D / 2 + D, src.cols - 1);
	int ty = min(2 * y - D / 2 + D, src.rows - 1);
	int cy = max(0, 2 * y - D / 2);
	float sum = 0;
	int count = 0;
	for (; cy < ty; ++cy) {
		for (int cx = max(0, 2 * x - D / 2); cx < tx; ++cx) {
			if (!isnan((float) src.ptr(cy)[cx])) {
				sum += src.ptr(cy)[cx]
						* kernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
				count += kernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
			}
		}
	}

	dst.ptr(y)[x] = (U) (sum / (float) count);
}

void PyrDownGaussian(const DeviceArray2D<float>& src,
		DeviceArray2D<float>& dst) {
	dim3 block(8, 8);
	dim3 grid(cv::divUp(dst.cols, block.x), cv::divUp(dst.rows, block.y));

	const float gaussKernel[25] = { 1, 4, 6, 4, 1, 4, 16, 24, 16, 4, 6, 24, 36,
			24, 6, 4, 16, 24, 16, 4, 1, 4, 6, 4, 1 };

	DeviceArray<float> kernel(25);
	kernel.upload((void*) gaussKernel, 25);

	PyrDownGaussian_device<float, float> <<<grid, block>>>(src, dst, kernel);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

void PyrDownGaussian(const DeviceArray2D<uchar>& src,
		DeviceArray2D<uchar>& dst) {
	dim3 block(8, 8);
	dim3 grid(cv::divUp(src.cols, block.x), cv::divUp(src.rows, block.y));

	const float gaussKernel[25] = { 1, 4, 6, 4, 1, 4, 16, 24, 16, 4, 6, 24, 36,
			24, 6, 4, 16, 24, 16, 4, 1, 4, 6, 4, 1 };

	DeviceArray<float> kernel(25);
	kernel.upload((void*) gaussKernel, 25);

	PyrDownGaussian_device<uchar, uchar> <<<grid, block>>>(src, dst, kernel);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

__global__ void ColourImageToIntensity_device(PtrStepSz<uchar3> src,
		PtrStep<uchar> dst) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= src.cols || y >= src.rows)
		return;

	uchar3 val = src.ptr(y)[x];
	int value = (float) val.x * 0.2126f + (float) val.y * 0.7152f
			+ (float) val.z * 0.0722f;
	dst.ptr(y)[x] = value;
}

void ColourImageToIntensity(const DeviceArray2D<uchar3>& src,
		DeviceArray2D<uchar>& dst) {
	dim3 block(8, 8);
	dim3 grid(cv::divUp(src.cols, block.x), cv::divUp(src.rows, block.y));

	ColourImageToIntensity_device<<<grid, block>>>(src, dst);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

__constant__ int sobely[9] = { 1, 2, 1, 0, 0, 0, -1, -2, -1 };
__constant__ int sobelx[9] = { 1, 0, -1, 2, 0, -2, 1, 0, -1 };

__global__ void ComputeDerivativeImage_device(PtrStepSz<uchar> src,
		PtrStep<float> dIx, PtrStep<float> dIy) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= src.cols || y >= src.rows)
		return;

	if (x > 0 && y > 0 && x < src.cols - 1 && y < src.rows - 1) {

		int dx = 0;
		int dy = 0;
		int id = 8;
		for (int i = -1; i < 2; ++i)
			for (int j = -1; j < 2; ++j) {
				int val = src.ptr(y + i)[x + j];
				dx += val * sobelx[id];
				dy += val * sobely[id];
				--id;
			}
		dIx.ptr(y)[x] = (float) dx / 8;
		dIy.ptr(y)[x] = (float) dy / 8;
	} else {
		dIx.ptr(y)[x] = 0;
		dIy.ptr(y)[x] = 0;
	}
}

void ComputeDerivativeImage(const DeviceArray2D<uchar>& src,
		DeviceArray2D<float>& dx, DeviceArray2D<float>& dy) {
	dim3 block(8, 8);
	dim3 grid(cv::divUp(src.cols, block.x), cv::divUp(src.rows, block.y));

	ComputeDerivativeImage_device<<<grid, block>>>(src, dx, dy);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

__global__ void ResizeMap_device(const PtrStepSz<float4> vsrc,
		const PtrStep<float4> nsrc, PtrStepSz<float4> vdst,
		PtrStep<float4> ndst) {

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x >= vsrc.cols || y >= vsrc.rows)
		return;

	float4 v00 = vsrc.ptr(y * 2 + 0)[x * 2 + 0];
	float4 v01 = vsrc.ptr(y * 2 + 0)[x * 2 + 1];
	float4 v10 = vsrc.ptr(y * 2 + 1)[x * 2 + 0];
	float4 v11 = vsrc.ptr(y * 2 + 1)[x * 2 + 1];
	float4 n00 = nsrc.ptr(y * 2 + 0)[x * 2 + 0];
	float4 n01 = nsrc.ptr(y * 2 + 0)[x * 2 + 1];
	float4 n10 = nsrc.ptr(y * 2 + 1)[x * 2 + 0];
	float4 n11 = nsrc.ptr(y * 2 + 1)[x * 2 + 1];

	if (isnan(v00.x) || isnan(v01.x) || isnan(v10.x) || isnan(v11.x)) {
		vdst.ptr(y)[x] = make_float4(__int_as_float(0x7fffffff));
	} else {
		vdst.ptr(y)[x] = (v00 + v01 + v10 + v11) / 4;
	}

	if (isnan(n00.x) || isnan(n01.x) || isnan(n10.x) || isnan(n11.x)) {
		ndst.ptr(y)[x] = make_float4(__int_as_float(0x7fffffff));
	} else {
		ndst.ptr(y)[x] = normalised((n00 + n01 + n10 + n11) / 4);
	}
}

void ResizeMap(const DeviceArray2D<float4>& vsrc,
		const DeviceArray2D<float4>& nsrc, DeviceArray2D<float4>& vdst,
		DeviceArray2D<float4>& ndst) {

	dim3 block(8, 8);
	dim3 grid(cv::divUp(vdst.cols, block.x), cv::divUp(vdst.rows, block.y));

	ResizeMap_device<<<grid, block>>>(vsrc, nsrc, vdst, ndst);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

__global__ void BackProjectPointsDevice(const PtrStepSz<float> src,
		PtrStepSz<float4> dst, float depthCutoff, float invfx, float invfy,
		float cx, float cy) {

	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= src.cols || y >= src.rows)
		return;

	float4 point;

	point.z = src.ptr(y)[x];
	if (!isnan(point.z) && point.z > 1e-3) {
		point.x = point.z * (x - cx) * invfx;
		point.y = point.z * (y - cy) * invfy;
		point.w = 1.0;
	}
	else {
		dst.ptr(y)[x] = make_float4(__int_as_float(0x7fffffff));
	}

	dst.ptr(y)[x] = point;
}

void BackProjectPoints(const DeviceArray2D<float>& src,
		DeviceArray2D<float4>& dst, float depthCutoff, float fx, float fy,
		float cx, float cy) {

	dim3 block(8, 8);
	dim3 grid(cv::divUp(src.cols, block.x), cv::divUp(src.rows, block.y));

	BackProjectPointsDevice<<<grid, block>>>(src, dst, depthCutoff, 1.0 / fx,
			1.0 / fy, cx, cy);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

__global__ void ComputeNormalMapDevice(const PtrStepSz<float4> src,
		PtrStepSz<float4> dst) {

	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= src.cols || y >= src.rows)
		return;

	if (x == src.cols - 1 || y == src.rows - 1) {
		dst.ptr(y)[x] = make_float4(__int_as_float(0x7fffffff));
		return;
	}

	float4 vcentre = src.ptr(y)[x];
	float4 vright = src.ptr(y)[x + 1];
	float4 vdown = src.ptr(y + 1)[x];

	if (!isnan(vcentre.x) && !isnan(vright.x) && !isnan(vdown.x)) {
		dst.ptr(y)[x] = normalised(cross(vright - vcentre, vdown - vcentre));
	} else
		dst.ptr(y)[x] = make_float4(__int_as_float(0x7fffffff));
}

void ComputeNormalMap(const DeviceArray2D<float4>& src,
		DeviceArray2D<float4>& dst) {

	dim3 block(8, 8);
	dim3 grid(cv::divUp(src.cols, block.x), cv::divUp(src.rows, block.y));

	ComputeNormalMapDevice<<<grid, block>>>(src, dst);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

__global__ void forwardProjectKernel(PtrStepSz<float4> src_vmap,
								     PtrStep<float4> src_nmap,
								     PtrStep<float4> dst_vmap,
								     PtrStep<float4> dst_nmap,
								     Matrix3f KRKinv, float3 Kt,
								     float fx, float fy,
								     float cx, float cy,
								     int cols, int rows) {

	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= src_vmap.cols || y >= src_vmap.rows)
		return;

	float3 pixel = make_float3(x, y, 1.f);
	pixel = KRKinv * pixel + Kt;
	int u = __float2int_rd(pixel.x / pixel.z * fx + cx + 0.5);
	int v = __float2int_rd(pixel.y / pixel.z * fy + cy + 0.5);
	if(u < 0 || v < 0 || u >= cols || v >= rows) {
		dst_vmap.ptr(y)[x] = make_float4(__int_as_float(0x7fffffff));
		dst_nmap.ptr(y)[x] = make_float4(__int_as_float(0x7fffffff));
		return;
	}

	dst_vmap.ptr(y)[x] = src_vmap.ptr(v)[u];
	dst_nmap.ptr(y)[x] = src_nmap.ptr(v)[u];
}

void forwardProjection(const DeviceArray2D<float4> & vsrc,
					   const DeviceArray2D<float4> & nsrc,
					   DeviceArray2D<float4> & vdst,
					   DeviceArray2D<float4> & ndst,
					   Matrix3f KRKinv, float3 Kt,
					   float fx, float fy,
					   float cx, float cy) {

	dim3 thread(16, 8);
	dim3 block(cv::divUp(vsrc.cols, thread.x), cv::divUp(vsrc.rows, thread.y));

//	forwardProjectKernel<<<block, thread>>>(vsrc, nsrc, vdst, ndst, Rcurr,
//			tcurr, RlastInv, tlast, fx, fy, cx, cy);
}

__global__ void RenderImageDevice(const PtrStep<float4> vmap,
								  const PtrStep<float4> nmap,
								  const float3 lightPose,
								  PtrStepSz<uchar4> dst) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= dst.cols || y >= dst.rows)
		return;

	float3 color;
	float3 p = make_float3(vmap.ptr(y)[x]);
	if (isnan(p.x)) {
		const float3 bgr1 = make_float3(4.f / 255.f, 2.f / 255.f, 2.f / 255.f);
		const float3 bgr2 = make_float3(236.f / 255.f, 120.f / 255.f,
				120.f / 255.f);

		float w = static_cast<float>(y) / dst.rows;
		color = bgr1 * (1 - w) + bgr2 * w;
	} else {
		float3 P = p;
		float3 N = make_float3(nmap.ptr(y)[x]);

		const float Ka = 0.3f;  //ambient coeff
		const float Kd = 0.5f;  //diffuse coeff
		const float Ks = 0.2f;  //specular coeff
		const float n = 20.f;  //specular power

		const float Ax = 1.f;   //ambient color,  can be RGB
		const float Dx = 1.f;   //diffuse color,  can be RGB
		const float Sx = 1.f;   //specular color, can be RGB
		const float Lx = 1.f;   //light color

		float3 L = normalised(lightPose - P);
		float3 V = normalised(make_float3(0.f, 0.f, 0.f) - P);
		float3 R = normalised(2 * N * (N * L) - L);

		float Ix = Ax * Ka * Dx + Lx * Kd * Dx * fmax(0.f, (N * L))
				+ Lx * Ks * Sx * __powf(fmax(0.f, (R * V)), n);
		color = make_float3(Ix, Ix, Ix);
	}

	uchar4 out;
	out.x = static_cast<unsigned char>(__saturatef(color.x) * 255.f);
	out.y = static_cast<unsigned char>(__saturatef(color.y) * 255.f);
	out.z = static_cast<unsigned char>(__saturatef(color.z) * 255.f);
	out.w = 255.0;
	dst.ptr(y)[x] = out;
}

void RenderImage(const DeviceArray2D<float4> & points,
				 const DeviceArray2D<float4> & normals,
				 const float3 light_pose,
				 DeviceArray2D<uchar4> & image) {

	dim3 block(8, 4);
	dim3 grid(cv::divUp(points.cols, block.x),
			  cv::divUp(points.rows, block.y));

	RenderImageDevice<<<grid, block>>>(points, normals, light_pose, image);

	SafeCall(hipGetLastError());
	SafeCall(hipDeviceSynchronize());
}

__global__ void depthToImageKernel(PtrStepSz<float> depth, PtrStepSz<uchar4> image) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= image.cols || y >= image.rows)
		return;

	float dp = depth.ptr(y)[x] / 20.0;
	int intdp = __float2int_rd(dp * 255);
	intdp = intdp > 255 ? 255 : intdp;
	image.ptr(y)[x] = make_uchar4(intdp, intdp, intdp, 255);
}

void depthToImage(const DeviceArray2D<float> & depth,
				  DeviceArray2D<uchar4> & image) {
	dim3 block(32, 8);
	dim3 grid(cv::divUp(image.cols, block.x),
			  cv::divUp(image.rows, block.y));

	depthToImageKernel<<<grid, block>>>(depth, image);

	SafeCall(hipGetLastError());
	SafeCall(hipDeviceSynchronize());
}

__global__ void rgbImageToRgbaKernel(PtrStepSz<uchar3> image, PtrStepSz<uchar4> rgba) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= image.cols || y >= image.rows)
		return;

	uchar3 rgb = image.ptr(y)[x];
	rgba.ptr(y)[x] = make_uchar4(rgb.x, rgb.y, rgb.z, 255);
}

void rgbImageToRgba(const DeviceArray2D<uchar3> & image,
				    DeviceArray2D<uchar4> & rgba) {
	dim3 block(32, 8);
	dim3 grid(cv::divUp(image.cols, block.x),
			  cv::divUp(image.rows, block.y));

	rgbImageToRgbaKernel<<<grid, block>>>(image, rgba);

	SafeCall(hipGetLastError());
	SafeCall(hipDeviceSynchronize());
}
