#include "hip/hip_runtime.h"
#include "DeviceMap.h"

#include <opencv.hpp>
#include <cudaarithm.hpp>

__device__ __forceinline__ float clamp(float a, float min = -1, float max = 1) {
	a = a > min ? a : min;
	a = a < max ? a : max;
	return a;
}

__global__ void BuildAdjecencyMatrixKernel(
		cv::cuda::PtrStepSz<float> adjecencyMatrix, PtrSz<SURF> frameKeys,
		PtrSz<SURF> mapKeys, PtrSz<float> dist) {

	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= adjecencyMatrix.cols || y >= adjecencyMatrix.rows)
		return;

	float score = 0;
//	if(x == y) {
//		score = exp(-dist[x]);
//	} else {

		SURF * mapKey00 = &mapKeys[x];
		SURF * mapKey01 = &mapKeys[y];

		SURF * frameKey00 = &frameKeys[x];
		SURF * frameKey01 = &frameKeys[y];

		float d00 = norm(frameKey00->pos - mapKey00->pos);
		float d01 = norm(frameKey01->pos - mapKey01->pos);

		float4 d10 = make_float4(frameKey00->pos - mapKey00->pos) / d00;
		float4 d11 = make_float4(frameKey01->pos - mapKey01->pos) / d01;

		if(d00 <= 1e-2 || d01 <= 1e-2)
			score = 0;

		float alpha00 = acos(clamp(frameKey00->normal * mapKey00->normal));
		float beta00 = acos(clamp(d10 * frameKey00->normal));
		float gamma00 = acos(clamp(d10 * mapKey00->normal));

		float alpha01 = acos(clamp(frameKey01->normal * mapKey01->normal));
		float beta01 = acos(clamp(d11 * frameKey01->normal));
		float gamma01 = acos(clamp(d11 * mapKey01->normal));

		score = exp(-(fabs(d00 - d01) + fabs(alpha00 - alpha01) + fabs(beta00 - beta01) + fabs(gamma00 - gamma01)));
//	}

	if(isnan(score))
		score = 0;

	adjecencyMatrix.ptr(y)[x] = score;
}

void BuildAdjecencyMatrix(cv::cuda::GpuMat & adjecencyMatrix,
		DeviceArray<SURF> & frameKeys, DeviceArray<SURF> & mapKeys,
		DeviceArray<float> & dist) {

	int cols = adjecencyMatrix.cols;
	int rows = adjecencyMatrix.rows;

	dim3 thread(8, 8);
	dim3 block(DivUp(cols, thread.x), DivUp(rows, thread.y));

	BuildAdjecencyMatrixKernel<<<block, thread>>>(adjecencyMatrix, frameKeys, mapKeys, dist);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());

	cv::cuda::GpuMat result;
	cv::cuda::reduce(adjecencyMatrix, result, 0, CV_REDUCE_SUM);
}

__global__ void FilterKeyMatchingKernel(PtrSz<SURF> trainKeys,
		PtrSz<SURF> queryKeys, PtrSz<SURF> trainKeysFiltered,
		PtrSz<SURF> queryKeysFiltered, PtrSz<int> matchesFiltered,
		PtrSz<int> queryIdx, PtrSz<int> keyIdxFiltered) {

	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x < matchesFiltered.size) {

		int idx = matchesFiltered[x];
		SURF * trainKey = &trainKeys[idx];
		SURF * queryKey = &queryKeys[idx];

		memcpy((void*) &trainKeysFiltered[x], (void*) trainKey, sizeof(SURF));
		memcpy((void*) &queryKeysFiltered[x], (void*) queryKey, sizeof(SURF));

		keyIdxFiltered[x] = queryIdx[idx];
	}
}

__global__ void ApplyContraintKernel(PtrStepSz<float> am, PtrSz<int> idx,
		PtrSz<int> flag) {

	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	if(x == y || x >= flag.size || y >= flag.size)
		return;

	int first = idx[x + 1];
	int second = idx[y + 1];
	if(am.ptr(first)[second] < 0.001f || am.ptr(second)[first] < 0.001f) {
		int headIdx = idx[0];
		if(am.ptr(first)[headIdx] < am.ptr(second)[headIdx])
			flag[x + 1] = 0;
	}
}

void FilterKeyMatching(cv::cuda::GpuMat & adjecencyMatrix,
		DeviceArray<SURF> & trainKey, DeviceArray<SURF> & queryKey,
		DeviceArray<SURF> & trainKeyFiltered,
		DeviceArray<SURF> & queryKeyFiltered, DeviceArray<int> & QueryIdx,
		DeviceArray<int> & keyIdxFiltered) {

	cv::cuda::GpuMat result;
	cv::cuda::reduce(adjecencyMatrix, result, 0, CV_REDUCE_SUM);
	cv::Mat cpuResult, indexMat;
	result.download(cpuResult);

	cv::sortIdx(cpuResult, indexMat, CV_SORT_EVERY_ROW + CV_SORT_DESCENDING);
	int selection = indexMat.cols >= 400 ? 400 : indexMat.cols;

//	cv::Mat am_cpu(adjecencyMatrix);
//	std::vector<cv::Mat> vmSelectedIdx;
//	cv::Mat cvNoSelected;
//	for(int i = 0; i < 10; ++i) {
//
//		cv::Mat mSelectedIdx;
//		int headIdx = 0;
//		int nSelected = 0;
//
//		for(int j = i; j < indexMat.cols; ++j) {
//
//			int idx = indexMat.at<int>(j);
//			if(nSelected == 0) {
//				mSelectedIdx.push_back(idx);
//				headIdx = idx;
//				nSelected++;
//			} else {
//				float score = am_cpu.at<float>(headIdx, idx);
//				if(score > 0.1f) {
//					mSelectedIdx.push_back(idx);
//					nSelected++;
//				}
//			}
//
//			if(nSelected >= 100)
//				break;
//		}
//
//		if(nSelected >= 4) {
//			cvNoSelected.push_back(nSelected);
//			vmSelectedIdx.push_back(mSelectedIdx);
//		}
//	}
//
//	cv::Mat tmp;
//	cv::sortIdx(cvNoSelected, tmp, CV_SORT_DESCENDING);
//	indexMat = vmSelectedIdx[tmp.at<int>(0)];
//	selection = indexMat.cols;
//	std::cout << indexMat << std::endl;

	DeviceArray<int> matchFiltered(selection);
	matchFiltered.upload((void*) indexMat.data, selection);

	trainKeyFiltered.create(selection);
	queryKeyFiltered.create(selection);
	keyIdxFiltered.create(selection);

	dim3 thread(MaxThread);
	dim3 block(DivUp(selection, thread.x));

	FilterKeyMatchingKernel<<<block, thread>>>(trainKey, queryKey,
			trainKeyFiltered, queryKeyFiltered, matchFiltered, QueryIdx,
			keyIdxFiltered);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}
