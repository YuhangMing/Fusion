#include "hip/hip_runtime.h"
#include "Timer.hpp"
#include "device_mapping.cuh"

__global__ void CollectORBKeys(KeyMap Km, PtrSz<ORBKey> index, int* totalKeys) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= Km.Keys.size)
		return;

	ORBKey* key = &Km.Keys[idx];

	if (key->valid) {
		int id = atomicAdd(totalKeys, 1);
		memcpy((void*) &index[id], (void*) key, sizeof(ORBKey));
	}
}

void CollectKeys(KeyMap Km, DeviceArray<ORBKey>& keys, int& n) {

	keys.create(Km.Keys.size);

	dim3 block(MaxThread);
	dim3 grid(cv::divUp(Km.Keys.size, block.x));

	DeviceArray<int> totalKeys(1);
	totalKeys.zero();

	CollectORBKeys<<<grid, block>>>(Km, keys, totalKeys);

	totalKeys.download(&n);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

__global__ void InsertKeysKernel(KeyMap map, PtrSz<ORBKey> key) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= key.size)
		return;

	map.InsertKey(&key[idx]);
}

void InsertKeys(KeyMap map, DeviceArray<ORBKey>& keys) {
	if (keys.size() == 0)
		return;

	dim3 block(MaxThread);
	dim3 grid(cv::divUp(keys.size(), block.x));

	InsertKeysKernel<<<grid, block>>>(map, keys);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

__global__ void ResetKeysKernel(KeyMap map) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	map.ResetKeys(idx);
}

void ResetKeys(KeyMap map) {
	dim3 block(MaxThread);
	dim3 grid(cv::divUp(map.MaxKeys * map.nBuckets, block.x));

	ResetKeysKernel<<<grid, block>>>(map);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}

__global__ void BuildAdjecencyMatrixKernel(PtrStepSz<float> AM,
		PtrSz<ORBKey> TrainKeys, PtrSz<ORBKey> QueryKeys,
		PtrSz<float> MatchDist) {

	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < AM.cols && y < AM.rows) {
		float score = 0;
		if(x == y) {
			score = expf(-MatchDist[x]);
		}
		else {
			ORBKey* match_0_train = &TrainKeys[x];
			ORBKey* match_0_query = &QueryKeys[x];
			ORBKey* match_1_train = &TrainKeys[y];
			ORBKey* match_1_query = &QueryKeys[y];
			float d_0 = norm(match_0_train->pos - match_0_query->pos);
			float d_1 = norm(match_1_train->pos - match_1_query->pos);
			if(d_0 > 1e-6 && d_1 > 1e-6) {
				float alpha_0 = acosf(match_0_train->normal * match_0_query->normal);
				float alpha_1 = acosf(match_1_train->normal * match_1_query->normal);
				float beta_0 = acosf(match_0_train->normal * (match_0_query->pos - match_0_train->pos));
				float beta_1 = acosf(match_1_train->normal * (match_1_query->pos - match_1_train->pos));
				float gamma_0 = acosf(match_0_query->normal * (match_0_train->pos - match_0_query->pos));
				float gamma_1 = acosf(match_1_query->normal * (match_1_train->pos - match_1_query->pos));
				score = expf(-(fabs(d_0 - d_1) + fabs(alpha_0 - alpha_1) + fabs(beta_0 - beta_1) + fabs(gamma_0 - gamma_1)));
			}
		}
		AM.ptr(y)[x] = score;
	}
}

void BuildAdjecencyMatrix(DeviceArray2D<float>& AM,	DeviceArray<ORBKey>& TrainKeys,
		DeviceArray<ORBKey>& QueryKeys, DeviceArray<float>& MatchDist) {

	dim3 block(32, 8);
	dim3 grid(cv::divUp(AM.cols(), block.x), cv::divUp(AM.rows(), block.y));

	BuildAdjecencyMatrixKernel<<<grid, block>>>(AM, TrainKeys, QueryKeys, MatchDist);

	cv::Mat test(AM.cols(), AM.cols(), CV_32FC1);
	AM.download((void*)test.data, test.step);
	std::cout << test << std::endl;
	cv::waitKey(0);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());
}
