#include "hip/hip_runtime.h"
#include "reduction.h"

__device__ __inline__ JtJJtrSE3 WarpReduceSum(JtJJtrSE3 val) {
#pragma unroll
	for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        val.aa += __shfl_down_sync(0xffffffff, val.aa, offset);
        val.ab += __shfl_down_sync(0xffffffff, val.ab, offset);
        val.ac += __shfl_down_sync(0xffffffff, val.ac, offset);
        val.ad += __shfl_down_sync(0xffffffff, val.ad, offset);
        val.ae += __shfl_down_sync(0xffffffff, val.ae, offset);
        val.af += __shfl_down_sync(0xffffffff, val.af, offset);
        val.ag += __shfl_down_sync(0xffffffff, val.ag, offset);

        val.bb += __shfl_down_sync(0xffffffff, val.bb, offset);
        val.bc += __shfl_down_sync(0xffffffff, val.bc, offset);
        val.bd += __shfl_down_sync(0xffffffff, val.bd, offset);
        val.be += __shfl_down_sync(0xffffffff, val.be, offset);
        val.bf += __shfl_down_sync(0xffffffff, val.bf, offset);
        val.bg += __shfl_down_sync(0xffffffff, val.bg, offset);

        val.cc += __shfl_down_sync(0xffffffff, val.cc, offset);
        val.cd += __shfl_down_sync(0xffffffff, val.cd, offset);
        val.ce += __shfl_down_sync(0xffffffff, val.ce, offset);
        val.cf += __shfl_down_sync(0xffffffff, val.cf, offset);
        val.cg += __shfl_down_sync(0xffffffff, val.cg, offset);

        val.dd += __shfl_down_sync(0xffffffff, val.dd, offset);
        val.de += __shfl_down_sync(0xffffffff, val.de, offset);
        val.df += __shfl_down_sync(0xffffffff, val.df, offset);
        val.dg += __shfl_down_sync(0xffffffff, val.dg, offset);

        val.ee += __shfl_down_sync(0xffffffff, val.ee, offset);
        val.ef += __shfl_down_sync(0xffffffff, val.ef, offset);
        val.eg += __shfl_down_sync(0xffffffff, val.eg, offset);

        val.ff += __shfl_down_sync(0xffffffff, val.ff, offset);
        val.fg += __shfl_down_sync(0xffffffff, val.fg, offset);

        val.residual += __shfl_down_sync(0xffffffff, val.residual, offset);
        val.inliers += __shfl_down_sync(0xffffffff, val.inliers, offset);
    }
    return val;
}

__device__ __inline__ JtJJtrSE3 BlockReduceSum(JtJJtrSE3 val) {

    static __shared__ JtJJtrSE3 shared[32];
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;
    val = WarpReduceSum(val);

	if (lane == 0) {
		shared[wid] = val;
	}
    __syncthreads();

    const JtJJtrSE3 zero = { 0, 0, 0, 0, 0, 0, 0, 0,
                             0, 0, 0, 0, 0, 0, 0, 0,
                             0, 0, 0, 0, 0, 0, 0, 0,
                             0, 0, 0, 0, 0};

	val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : zero;
	if (wid == 0) {
		val = WarpReduceSum(val);
	}

    return val;
}

__global__ void ReduceSum(JtJJtrSE3 * in, JtJJtrSE3 * out, int N) {

    JtJJtrSE3 sum = { 0, 0, 0, 0, 0, 0, 0, 0,
                      0, 0, 0, 0, 0, 0, 0, 0,
                      0, 0, 0, 0, 0, 0, 0, 0,
                      0, 0, 0, 0, 0};

    int i = blockIdx.x * blockDim.x + threadIdx.x;
	for (; i < N; i += blockDim.x * gridDim.x) {
        sum.add(in[i]);
    }

    sum = BlockReduceSum(sum);

	if (threadIdx.x == 0) {
        out[blockIdx.x] = sum;
    }
}

struct ICPReduction {

	PtrStep<float4> nextVMap;
	PtrStep<float4> lastVMap;
	PtrStep<float3> nextNMap;
	PtrStep<float3> lastNMap;

	Matrix3f Rcurr;
	Matrix3f Rlast;
	Matrix3f RlastInv;
	float3 tcurr;
	float3 tlast;

	MatK K;
	float angleThresh;
	float distThresh;
	int cols, rows, N;

	JtJJtrSE3 * out;

	__device__ inline bool findCorresp(int & x, int & y,
								       float3 & vcurr_g,
								       float3 & vlast_g,
								       float3 & nlast_g) const {

		float3 vcurr_c = make_float3(nextVMap.ptr(y)[x]);
		if (isnan(vcurr_c.x) || vcurr_c.z < 1e-3)
			return false;

		vcurr_g = Rcurr * vcurr_c + tcurr;
		float3 vcurr_p = RlastInv * (vcurr_g - tlast);

		float invz = 1.0 / vcurr_p.z;
		int u = (int) (vcurr_p.x * invz * K.fx + K.cx);
		int v = (int) (vcurr_p.y * invz * K.fy + K.cy);
		if (u < 0 || v < 0 || u >= cols || v >= rows)
			return false;

		float3 vlast_c = make_float3(lastVMap.ptr(v)[u]);
		vlast_g = Rlast * vlast_c + tlast;

		float3 ncurr_c = nextNMap.ptr(y)[x];
		float3 ncurr_g = Rcurr * ncurr_c;

		float3 nlast_c = lastNMap.ptr(v)[u];
		nlast_g = Rlast * nlast_c;

		float dist = norm(vlast_g - vcurr_g);
		float sine = norm(cross(ncurr_g, nlast_g));

		return (sine < angleThresh &&
				dist <= distThresh &&
				!isnan(ncurr_c.x) &&
				!isnan(nlast_c.x));
	}

	__device__ __inline__ JtJJtrSE3 getProduct(int & k) const {

		int y = k / cols;
		int x = k - (y * cols);
		float3 vcurr, vlast, nlast;
		bool found_coresp = findCorresp(x, y, vcurr, vlast, nlast);

		float row[7] = { 0, 0, 0, 0, 0, 0, 0 };
		if (found_coresp) {
			nlast = RlastInv * nlast;
			vcurr = RlastInv * (vcurr - tlast);
			vlast = RlastInv * (vlast - tlast);
			*(float3*) &row[0] = -nlast;
			*(float3*) &row[3] = cross(nlast, vlast);
			row[6] = -nlast * (vlast - vcurr);
		}

        JtJJtrSE3 val = { row[0] * row[0],
                          row[0] * row[1],
                          row[0] * row[2],
                          row[0] * row[3],
                          row[0] * row[4],
                          row[0] * row[5],
                          row[0] * row[6],

                          row[1] * row[1],
                          row[1] * row[2],
                          row[1] * row[3],
                          row[1] * row[4],
                          row[1] * row[5],
                          row[1] * row[6],

                          row[2] * row[2],
                          row[2] * row[3],
                          row[2] * row[4],
                          row[2] * row[5],
                          row[2] * row[6],

                          row[3] * row[3],
                          row[3] * row[4],
                          row[3] * row[5],
                          row[3] * row[6],

                          row[4] * row[4],
                          row[4] * row[5],
                          row[4] * row[6],

                          row[5] * row[5],
                          row[5] * row[6],

                          row[6] * row[6],
                          (float)found_coresp};
        return val;
	}

	__device__ void operator()() const {

	    JtJJtrSE3 sum = { 0, 0, 0, 0, 0, 0, 0, 0,
	                      0, 0, 0, 0, 0, 0, 0, 0,
	                      0, 0, 0, 0, 0, 0, 0, 0,
	                      0, 0, 0, 0, 0};

		int i = blockIdx.x * blockDim.x + threadIdx.x;
		for (; i < N; i += blockDim.x * gridDim.x) {
			JtJJtrSE3 val = getProduct(i);
			sum.add(val);
		}

		BlockReduceSum(sum);

		if (threadIdx.x == 0)
			out[blockIdx.x] = sum;
	}
};

__global__ void icpStepKernel(const ICPReduction icp) {
	icp();
}

void icpStep(const DeviceArray2D<float4> & nextVMap,
			 const DeviceArray2D<float4> & lastVMap,
			 const DeviceArray2D<float3> & nextNMap,
			 const DeviceArray2D<float3> & lastNMap,
			 DeviceArray<JtJJtrSE3> & sum,
			 DeviceArray<JtJJtrSE3> & out,
			 float * residual,
			 double * JtJ_host,
			 double * Jtr_host,
			 Matrix3f Rcurr,
			 float3 tcurr,
			 Matrix3f Rlast,
			 Matrix3f RlastInv,
			 float3 tlast,
			 MatK K) {

	int cols = nextVMap.cols();
	int rows = nextVMap.rows();

	ICPReduction icp;

	icp.K = K;
	icp.cols = cols;
	icp.rows = rows;
	icp.N = cols * rows;

	icp.Rcurr = Rcurr;
	icp.tcurr = tcurr;
	icp.Rlast = Rlast;
	icp.RlastInv = RlastInv;
	icp.tlast = tlast;

	icp.nextVMap = nextVMap;
	icp.lastVMap = lastVMap;
	icp.nextNMap = nextNMap;
	icp.lastNMap = lastNMap;

	icp.angleThresh = 0.6;
	icp.distThresh = 0.1;

	icp.out = sum;

	icpStepKernel<<<96, 224>>>(icp);

	ReduceSum<<<1, 512>>>(sum, out, 96);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());

	float host_data[29];
	out.download((JtJJtrSE3*) host_data);

	int shift = 0;
	for (int i = 0; i < 6; ++i) {
		for (int j = i; j < 7; ++j) {
			double value = (double)host_data[shift++];
			if (j == 6)
				Jtr_host[i] = value;
			else
				JtJ_host[j * 6 + i] = JtJ_host[i * 6 + j] = value;
		}
	}

	residual[0] = host_data[27];
	residual[1] = host_data[28];
}

template<typename T, int size> __device__
inline void WarpReduceSum(T* val) {
	for (int offset = WarpSize / 2; offset > 0; offset /= 2) {
#pragma unroll
		for (int i = 0; i < size; ++i) {
			val[i] += __shfl_down_sync(0xffffffff, val[i], offset);
		}
	}
}

template<typename T, int size> __device__
inline void BlockReduceSum(T* val) {
	static __shared__ T shared[32 * size];
	int lane = threadIdx.x % warpSize;
	int wid = threadIdx.x / warpSize;

	WarpReduceSum<T, size>(val);

	if (lane == 0)
		memcpy(&shared[wid * size], val, sizeof(T) * size);

	__syncthreads();

	if (threadIdx.x < blockDim.x / warpSize)
		memcpy(val, &shared[lane * size], sizeof(T) * size);
	else
		memset(val, 0, sizeof(T) * size);

	if (wid == 0)
		WarpReduceSum<T, size>(val);
}

template<typename T, int size> __global__
void ReduceSum(PtrStep<T> in, T* out, int N) {
	T sum[size];
	memset(sum, 0, sizeof(T) * size);
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	for (; i < N; i += blockDim.x * gridDim.x)
#pragma unroll
		for (int j = 0; j < size; ++j)
			sum[j] += in.ptr(i)[j];

	BlockReduceSum<T, size>(sum);

	if (threadIdx.x == 0)
#pragma unroll
		for (int i = 0; i < size; ++i)
			out[i] = sum[i];
}

struct ICPReduce {

	Matrix3f Rcurr;
	Matrix3f Rlast;
	Matrix3f invRlast;
	float3 tcurr;
	float3 tlast;
	PtrStep<float4> VMapCurr, VMapLast;
	PtrStep<float3> NMapCurr, NMapLast;
	int cols, rows, N;
	float fx, fy, cx, cy;
	float angleThresh, distThresh;

	mutable PtrStepSz<float> out;

	__device__ inline
	bool SearchPoint(int& x, int& y, float3& vcurr_g, float3& vlast_g,
			float3& nlast_g) const {

		float3 vcurr_c = make_float3(VMapCurr.ptr(y)[x]);
		if (isnan(vcurr_c.x) || vcurr_c.z < 1e-3)
			return false;

		vcurr_g = Rcurr * vcurr_c + tcurr;
		float3 vcurr_p = invRlast * (vcurr_g - tlast);

		float invz = 1.0 / vcurr_p.z;
		int u = (int) (vcurr_p.x * invz * fx + cx + 0.5);
		int v = (int) (vcurr_p.y * invz * fy + cy + 0.5);
		if (u < 0 || v < 0 || u >= cols || v >= rows)
			return false;

		float3 vlast_c = make_float3(VMapLast.ptr(v)[u]);
		vlast_g = Rlast * vlast_c + tlast;

		float3 ncurr_c = NMapCurr.ptr(y)[x];
		float3 ncurr_g = Rcurr * ncurr_c;

		float3 nlast_c = NMapLast.ptr(v)[u];
		nlast_g = Rlast * nlast_c;

		float dist = norm(vlast_g - vcurr_g);
		float sine = norm(cross(ncurr_g, nlast_g));

		return (sine < angleThresh && dist <= distThresh && !isnan(ncurr_c.x)
				&& !isnan(nlast_c.x));
	}

	__device__ inline
	void GetRow(int& i, float* sum) const {
		int y = i / cols;
		int x = i - (y * cols);

		bool found = false;
		float3 vcurr, vlast, nlast;
		found = SearchPoint(x, y, vcurr, vlast, nlast);
		float row[7] = { 0, 0, 0, 0, 0, 0, 0 };

		if (found) {
			nlast = invRlast * nlast;
			vcurr = invRlast * (vcurr - tlast);
			vlast = invRlast * (vlast - tlast);
			*(float3*) &row[0] = -nlast;
			*(float3*) &row[3] = cross(nlast, vlast);
			row[6] = -nlast * (vlast - vcurr);
		}

		int count = 0;
#pragma unroll
		for (int i = 0; i < 7; ++i)
#pragma unroll
			for (int j = i; j < 7; ++j)
				sum[count++] = row[i] * row[j];

		sum[count] = (float) found;
	}

	template<typename T, int size>
	__device__ void operator()() const {
		T sum[size];
		T val[size];
		memset(sum, 0, sizeof(T) * size);
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		for (; i < N; i += blockDim.x * gridDim.x) {
			memset(val, 0, sizeof(T) * size);
			GetRow(i, val);
#pragma unroll
			for (int j = 0; j < size; ++j)
				sum[j] += val[j];
		}

		BlockReduceSum<T, size>(sum);

		if (threadIdx.x == 0)
#pragma unroll
			for (int i = 0; i < size; ++i)
				out.ptr(blockIdx.x)[i] = sum[i];
	}
};

__global__ void ICPReduceSum_device(const ICPReduce icp) {
	icp.template operator()<float, 29>();
}

static void inline CreateMatrix(float* host_data, double* host_a,
		double* host_b) {
	int shift = 0;
	for (int i = 0; i < 6; ++i)
		for (int j = i; j < 7; ++j) {
			double value = (double)host_data[shift++];
			if (j == 6)
				host_b[i] = value;
			else
				host_a[j * 6 + i] = host_a[i * 6 + j] = value;
		}
}

double ICPReduceSum(DeviceArray2D<float4> & nextVMap, DeviceArray2D<float4> & lastVMap,
		DeviceArray2D<float3> & nextNMap, DeviceArray2D<float3> & lastNMap,
		Frame& NextFrame, Frame& LastFrame, int pyr, double* host_a,
		double* host_b) {

	DeviceArray2D<float> sum(29, 96);
	DeviceArray<float> result(29);
	result.zero();
	sum.zero();

	ICPReduce icp;
	icp.out = sum;
	icp.VMapCurr = nextVMap;
	icp.NMapCurr = nextNMap;
	icp.VMapLast = lastVMap;
	icp.NMapLast = lastNMap;
	icp.cols = Frame::cols(pyr);
	icp.rows = Frame::rows(pyr);
	icp.N = Frame::cols(pyr) * Frame::rows(pyr);
	icp.Rcurr = NextFrame.Rot_gpu();
	icp.tcurr = NextFrame.Trans_gpu();
	icp.Rlast = LastFrame.Rot_gpu();
	icp.invRlast = LastFrame.RotInv_gpu();
	icp.tlast = LastFrame.Trans_gpu();
	icp.angleThresh = 0.6;
	icp.distThresh = 0.1;
	icp.fx = Frame::fx(pyr);
	icp.fy = Frame::fy(pyr);
	icp.cx = Frame::cx(pyr);
	icp.cy = Frame::cy(pyr);

	ICPReduceSum_device<<<96, 224>>>(icp);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());

	ReduceSum<float, 29> <<<1, MaxThread>>>(sum, result, 96);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());

	float host_data[29];
	result.download(host_data);
	CreateMatrix(host_data, host_a, host_b);
	return sqrt(host_data[27]) / host_data[28];
}
