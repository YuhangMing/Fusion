#include "hip/hip_runtime.h"
#include "device_math.hpp"

DEV_FUNC uchar3 make_uchar3(float3 a) {
	return make_uchar3(__float2int_rd(a.x),
					   __float2int_rd(a.y),
					   __float2int_rd(a.z));
}

HOST_FUNC DEV_FUNC int2 make_int2(int a) {
	return make_int2(a, a);
}

DEV_FUNC int2 make_int2(float2 a) {
	return make_int2(__float2int_rd(a.x),
				     __float2int_rd(a.y));
}

HOST_FUNC DEV_FUNC int3 make_int3(int a) {
	return make_int3(a, a, a);
}

DEV_FUNC int3 make_int3(float a) {
	return make_int3(make_float3(a));
}

DEV_FUNC int3 make_int3(float3 a) {
	return make_int3(__float2int_rd(a.x),
					 __float2int_rd(a.y),
					 __float2int_rd(a.z));
}

HOST_FUNC DEV_FUNC int4 make_int4(int3 a, int b) {
	return make_int4(a.x, a.y, a.z, b);
}


HOST_FUNC DEV_FUNC uint2 make_uint2(int a) {
	return make_uint2(a, a);
}

HOST_FUNC DEV_FUNC uint3 make_uint3(int a) {
	return make_uint3(a, a, a);
}

HOST_FUNC DEV_FUNC float2 make_float2(float a) {
	return make_float2(a, a);
}

HOST_FUNC DEV_FUNC float3 make_float3(uchar3 a) {
	return make_float3(a.x, a.y, a.z);
}

HOST_FUNC DEV_FUNC float3 make_float3(float a) {
	return make_float3(a, a, a);
}

HOST_FUNC DEV_FUNC float3 make_float3(int3 a) {
	return make_float3(a.x, a.y, a.z);
}

HOST_FUNC DEV_FUNC float3 make_float3(float4 a) {
	return make_float3(a.x, a.y, a.z);
}

HOST_FUNC DEV_FUNC float4 make_float4(float a) {
	return make_float4(a, a, a, a);
}

HOST_FUNC DEV_FUNC float4 make_float4(float3 a) {
	return make_float4(a.x, a.y, a.z, 1.f);
}

HOST_FUNC DEV_FUNC float4 make_float4(float3 a, float b) {
	return make_float4(a.x, a.y, a.z, b);
}

HOST_FUNC DEV_FUNC double4 make_double4(double a) {
	return make_double4(a, a, a, a);
}

HOST_FUNC DEV_FUNC int2 operator+(int2 a, int2 b) {
	return make_int2(a.x + b.x, a.y + b.y);
}

HOST_FUNC DEV_FUNC float2 operator+(float2 a, float2 b) {
	return make_float2(a.x + b.x, a.y + b.y);
}

HOST_FUNC DEV_FUNC uchar3 operator+(uchar3 a, uchar3 b) {
	return make_uchar3(a.x + b.x, a.y + b.y, a.z + b.z);
}

HOST_FUNC DEV_FUNC int3 operator+(int3 a, int3 b) {
	return make_int3(a.x + b.x, a.y + b.y, a.z + b.z);
}

HOST_FUNC DEV_FUNC float3 operator+(float3 a, float b) {
	return make_float3(a.x + b, a.y + b, a.z + b);
}

HOST_FUNC DEV_FUNC float3 operator+(float3 a, float3 b) {
	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

HOST_FUNC DEV_FUNC float4 operator+(float4 a, float4 b) {
	return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

HOST_FUNC DEV_FUNC void operator+=(float3 & a, uchar3 b) {
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
}

HOST_FUNC DEV_FUNC void operator+=(float3 & a, float3 b) {
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
}

HOST_FUNC DEV_FUNC void operator-=(float3 & a, float3 b) {
	a.x -= b.x;
	a.y -= b.y;
	a.z -= b.z;
}

HOST_FUNC DEV_FUNC int2 operator-(int2 a, int2 b) {
	return make_int2(a.x - b.x, a.y - b.y);
}

HOST_FUNC DEV_FUNC float3 operator-(float3 b) {
	return make_float3(-b.x, -b.y, -b.z);
}

HOST_FUNC DEV_FUNC float3 operator-(float3 a, float b) {
	return make_float3(a.x - b, a.y - b, a.z - b);
}

HOST_FUNC DEV_FUNC float3 operator-(float a, float3 b) {
	return make_float3(a - b.x, a - b.y, a - b.z);
}

HOST_FUNC DEV_FUNC float3 operator-(float3 a, float3 b) {
	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

HOST_FUNC DEV_FUNC float4 operator-(float4 a, float4 b) {
	return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w);
}

HOST_FUNC DEV_FUNC
float operator*(float3 a, float3 b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

HOST_FUNC DEV_FUNC
float operator*(float3 a, float4 b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

HOST_FUNC DEV_FUNC
float operator*(float4 a, float3 b) {
	return a.x * b.x + a.y * b.y + a.z * b.z + a.w;
}

HOST_FUNC DEV_FUNC
float operator*(float4 a, float4 b) {
	return a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w;
}

HOST_FUNC DEV_FUNC uchar3 operator*(uchar3 a, unsigned short b) {
	return make_uchar3(a.x * b, a.y * b, a.z * b);
}

HOST_FUNC DEV_FUNC uchar3 operator*(uchar3 a, int b) {
	return make_uchar3(a.x * b, a.y * b, a.z * b);
}

HOST_FUNC DEV_FUNC uchar3 operator*(int b, uchar3 a) {
	return make_uchar3(a.x * b, a.y * b, a.z * b);
}

HOST_FUNC DEV_FUNC int3 operator*(int3 a, uint b) {
	return make_int3(a.x * b, a.y * b, a.z * b);
}

HOST_FUNC DEV_FUNC int3 operator*(int3 a, int b) {
	return make_int3(a.x * b, a.y * b, a.z * b);
}

HOST_FUNC DEV_FUNC int3 operator*(float3 a, int b) {
	return make_int3(a.x * b, a.y * b, a.z * b);
}

HOST_FUNC DEV_FUNC float3 operator*(int3 a, float b) {
	return make_float3(a.x * b, a.y * b, a.z * b);
}

HOST_FUNC DEV_FUNC float3 operator*(float3 a, float b) {
	return make_float3(a.x * b, a.y * b, a.z * b);
}

HOST_FUNC DEV_FUNC float3 operator*(float a, float3 b) {
	return make_float3(a * b.x, a * b.y, a * b.z);
}

HOST_FUNC DEV_FUNC float4 operator*(float4 a, float b) {
	return make_float4(a.x * b, a.y * b, a.z * b, a.w * b);
}

HOST_FUNC DEV_FUNC int3 operator/(int3 a, int3 b) {
	return make_int3(a.x / b.x, a.y / b.y, a.z / b.z);
}

HOST_FUNC DEV_FUNC float3 operator/(float3 a, int3 b) {
	return make_float3(a.x / b.x, a.y / b.y, a.z / b.z);
}

HOST_FUNC DEV_FUNC float3 operator/(float3 a, float3 b) {
	return make_float3(a.x / b.x, a.y / b.y, a.z / b.z);
}

HOST_FUNC DEV_FUNC float4 operator/(float4 a, float4 b) {
	return make_float4(a.x / b.x, a.y / b.y, a.z / b.z, a.w / b.w);
}


HOST_FUNC DEV_FUNC int2 operator/(int2 a, int b) {
	return make_int2(a.x / b, a.y / b);
}

HOST_FUNC DEV_FUNC float2 operator/(float2 a, int b) {
	return make_float2(a.x / b, a.y / b);
}

HOST_FUNC DEV_FUNC uchar3 operator/(uchar3 a, int b) {
	return make_uchar3(a.x / b, a.y / b, a.z / b);
}

HOST_FUNC DEV_FUNC int3 operator/(int3 a, uint b) {
	return make_int3(a.x / (int) b, a.y / (int) b, a.z / (int) b);
}

HOST_FUNC DEV_FUNC int3 operator/(int3 a, int b) {
	return make_int3(a.x / b, a.y / b, a.z / b);
}

HOST_FUNC DEV_FUNC float3 operator/(float3 a, int b) {
	return make_float3(a.x / b, a.y / b, a.z / b);
}

HOST_FUNC DEV_FUNC float3 operator/(float3 a, float b) {
	return make_float3(a.x / b, a.y / b, a.z / b);
}

HOST_FUNC DEV_FUNC float3 operator/(float a, float3 b) {
	return make_float3(a / b.x, a / b.y, a / b.z);
}

HOST_FUNC DEV_FUNC float4 operator/(float4 a, float b) {
	return make_float4(a.x / b, a.y / b, a.z / b, a.w / b);
}

HOST_FUNC DEV_FUNC int3 operator%(int3 a, int b) {
	return make_int3(a.x % b, a.y % b, a.z % b);
}

HOST_FUNC DEV_FUNC
bool operator==(int3 a, int3 b) {
	return a.x == b.x && a.y == b.y && a.z == b.z;
}

DEV_FUNC float3 cross(float3 a, float3 b) {
	return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z,
			a.x * b.y - a.y * b.x);
}

DEV_FUNC float3 cross(float4 a, float4 b) {
	return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z,
			a.x * b.y - a.y * b.x);
}

DEV_FUNC void atomicMax(float* address, float val) {
	int* address_as_i = (int*) address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_i,
				assumed, __float_as_int(fmaxf(val, __int_as_float(assumed))));
	} while (assumed != old);
}

DEV_FUNC void atomicMin(float* address, float val) {
	int* address_as_i = (int*) address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_i,
				assumed, __float_as_int(fminf(val, __int_as_float(assumed))));
	} while (assumed != old);
}

DEV_FUNC
float norm(float3 a) {
	return sqrtf(a * a);
}

DEV_FUNC
float norm(float4 a) {
	return sqrtf(a * a);
}

DEV_FUNC
float inv_norm(float3 a) {
	return rsqrtf(a * a);
}

DEV_FUNC
float inv_norm(float4 a) {
	return rsqrtf(a * a);
}

DEV_FUNC float3 normalised(float3 a) {
	return a / norm(a);
}

DEV_FUNC float4 normalised(float4 a) {
	return a / norm(a);
}

DEV_FUNC float3 floor(float3 a) {
	return make_float3(floor(a.x), floor(a.y), floor(a.z));
}

HOST_FUNC DEV_FUNC Matrix3f::Matrix3f() {
	rowx = rowy = rowz = make_float3(0, 0, 0);
}

HOST_FUNC DEV_FUNC Matrix3f Matrix3f::Identity() {
	Matrix3f id;
	id.rowx = make_float3(1, 0, 0);
	id.rowy = make_float3(0, 1, 0);
	id.rowz = make_float3(0, 0, 1);
	return id;
}

HOST_FUNC DEV_FUNC float3 Matrix3f::operator*(float3 vec) {
	return make_float3(rowx * vec, rowy * vec, rowz * vec);
}

HOST_FUNC DEV_FUNC float3 Matrix3f::operator*(float3 vec) const {
	return make_float3(rowx * vec, rowy * vec, rowz * vec);
}

HOST_FUNC DEV_FUNC float4 Matrix3f::operator*(float4 vec) {
	return make_float4(rowx * vec, rowy * vec, rowz * vec, vec.w);
}

HOST_FUNC DEV_FUNC float4 Matrix3f::operator*(float4 vec) const {
	return make_float4(rowx * vec, rowy * vec, rowz * vec, vec.w);
}
