#include "hip/hip_runtime.h"
#include "Mapping.hpp"

template<int threadBlock>
DEV_FUNC int ComputeOffset(uint element, uint *sum) {

	__shared__ uint buffer[threadBlock];
	__shared__ uint blockOffset;

	if (threadIdx.x == 0)
		memset(buffer, 0, sizeof(uint) * 16 * 16);
	__syncthreads();

	buffer[threadIdx.x] = element;
	__syncthreads();

	int s1, s2;

	for (s1 = 1, s2 = 1; s1 < threadBlock; s1 <<= 1) {
		s2 |= s1;
		if ((threadIdx.x & s2) == s2)
			buffer[threadIdx.x] += buffer[threadIdx.x - s1];
		__syncthreads();
	}

	for (s1 >>= 2, s2 >>= 1; s1 >= 1; s1 >>= 1, s2 >>= 1) {
		if (threadIdx.x != threadBlock - 1 && (threadIdx.x & s2) == s2)
			buffer[threadIdx.x + s1] += buffer[threadIdx.x];
		__syncthreads();
	}

	if (threadIdx.x == 0 && buffer[threadBlock - 1] > 0)
		blockOffset = atomicAdd(sum, buffer[threadBlock - 1]);
	__syncthreads();

	int offset;
	if (threadIdx.x == 0) {
		if (buffer[threadIdx.x] == 0)
			offset = -1;
		else
			offset = blockOffset;
	} else {
		if (buffer[threadIdx.x] == buffer[threadIdx.x - 1])
			offset = -1;
		else
			offset = blockOffset + buffer[threadIdx.x - 1];
	}

	return offset;
}

struct HashMarchingCube {

	DeviceMap map;
	float3* triangles;
	PtrStep<int> triangleTable;
	PtrSz<int> edgeTable;
	PtrSz<int3> vPos;
	uint* noBlocks;
	uint* noTriangles;
	float2* thresh;
	Matrix3f Rot, invRot;
	float3 trans;
	uchar3* color;
	float3* normals;
	int cols, rows;
	float fx, fy, cx, cy;

	DEV_FUNC void FindExistingBlocks() {
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		__shared__ bool scan;
		if(x == 0)
			scan = false;
		__syncthreads();
		uint val = 0;
		if (x < DeviceMap::NumEntries) {
			if (map.hashEntries[x].ptr >= 0) {
				int3 pos = map.hashEntries[x].pos * DeviceMap::BlockSize;
//				if(map.FindVoxel(pos).GetSdf() < 0 ||
//				   map.FindVoxel(pos + make_int3(7, 0, 0)).GetSdf() < 0 ||
//			       map.FindVoxel(pos + make_int3(0, 7, 0)).GetSdf() < 0 ||
//				   map.FindVoxel(pos + make_int3(7, 0, 7)).GetSdf() < 0 ||
//				   map.FindVoxel(pos + make_int3(7, 7, 0)).GetSdf() < 0 ||
//				   map.FindVoxel(pos + make_int3(7, 0, 7)).GetSdf() < 0 ||
//				   map.FindVoxel(pos + make_int3(0, 7, 7)).GetSdf() < 0 ||
//				   map.FindVoxel(pos + make_int3(7, 7, 7)).GetSdf() < 0) {
					scan = true;
					val = 1;
//				}
			}
		}
		__syncthreads();
		if(scan) {
			int offset = ComputeOffset<1024>(val, noBlocks);
			if(offset != -1) {
				vPos[offset] = map.hashEntries[x].pos;
			}
		}
	}

	DEV_FUNC bool FindNormal(float3* n, float* sdf, int3 pos) {
		float v1, v2, v3;
		v1 = map.FindVoxel(pos + make_int3(-1, 0, 0)).GetSdf();
		v2 = map.FindVoxel(pos + make_int3(0, -1, 0)).GetSdf();
		v3 = map.FindVoxel(pos + make_int3(0, 0, -1)).GetSdf();
		if(isnan(v1) || isnan(v2) || isnan(v3))
			return false;
		n[0] = make_float3(sdf[1] - v1, sdf[3] - v2, sdf[4] - v3);

		v1 = map.FindVoxel(pos + make_int3(2, 0, 0)).GetSdf();
		v2 = map.FindVoxel(pos + make_int3(1, -1, 0)).GetSdf();
		v3 = map.FindVoxel(pos + make_int3(1, 0, -1)).GetSdf();
		if(isnan(v1) || isnan(v2) || isnan(v3))
			return false;
		n[1] = make_float3(v1 - sdf[0], sdf[2] - v2, sdf[5] - v3);

		v1 = map.FindVoxel(pos + make_int3(2, 1, 0)).GetSdf();
		v2 = map.FindVoxel(pos + make_int3(1, 2, 0)).GetSdf();
		v3 = map.FindVoxel(pos + make_int3(1, 1, -1)).GetSdf();
		if(isnan(v1) || isnan(v2) || isnan(v3))
			return false;
		n[2] = make_float3(v1 - sdf[3], v2 - sdf[1], sdf[6] - v3);

		v1 = map.FindVoxel(pos + make_int3(-1, 1, 0)).GetSdf();
		v2 = map.FindVoxel(pos + make_int3(0, 2, 0)).GetSdf();
		v3 = map.FindVoxel(pos + make_int3(0, 1, -1)).GetSdf();
		if(isnan(v1) || isnan(v2) || isnan(v3))
			return false;
		n[3] = make_float3(sdf[2] - v1, v2 - sdf[0], sdf[7] - v3);

		v1 = map.FindVoxel(pos + make_int3(-1, 0, 1)).GetSdf();
		v2 = map.FindVoxel(pos + make_int3(0, -1, 1)).GetSdf();
		v3 = map.FindVoxel(pos + make_int3(0, 0, 2)).GetSdf();
		if(isnan(v1) || isnan(v2) || isnan(v3))
			return false;
		n[4] = make_float3(sdf[5] - v1, sdf[7] - v2, v3 - sdf[0]);

		v1 = map.FindVoxel(pos + make_int3(2, 0, 1)).GetSdf();
		v2 = map.FindVoxel(pos + make_int3(1, -1, 1)).GetSdf();
		v3 = map.FindVoxel(pos + make_int3(1, 0, 2)).GetSdf();
		if(isnan(v1) || isnan(v2) || isnan(v3))
			return false;
		n[5] = make_float3(v1 - sdf[4], sdf[6] - v2 , v3 - sdf[1]);

		v1 = map.FindVoxel(pos + make_int3(2, 1, 1)).GetSdf();
		v2 = map.FindVoxel(pos + make_int3(1, 2, 1)).GetSdf();
		v3 = map.FindVoxel(pos + make_int3(1, 1, 2)).GetSdf();
		if(isnan(v1) || isnan(v2) || isnan(v3))
			return false;
		n[6] = make_float3(v1 - sdf[7], v2 - sdf[5] , v3 - sdf[2]);

		v1 = map.FindVoxel(pos + make_int3(-1, 1, 1)).GetSdf();
		v2 = map.FindVoxel(pos + make_int3(0, 2, 1)).GetSdf();
		v3 = map.FindVoxel(pos + make_int3(0, 1, 2)).GetSdf();
		if(isnan(v1) || isnan(v2) || isnan(v3))
			return false;
		n[7] = make_float3(sdf[6] - v1, v2 - sdf[4] , v3 - sdf[3]);

		return true;
	}

	DEV_FUNC bool FindPoint(float3* p, float3* n, uchar3* c, float* sdf, int3 pos) {

		int3 localPos;
		Voxel v;

		localPos = pos + make_int3(0, 0, 0);
		p[0] = make_float3(localPos);
		v = map.FindVoxel(localPos);
		sdf[0] = v.GetSdf();
		c[0] = v.rgb;
		if (sdf[0] == 1.0 || isnan(sdf[0]))
			return false;

		localPos = pos + make_int3(1, 0, 0);
		p[1] = make_float3(localPos);
		v = map.FindVoxel(localPos);
		sdf[1] = v.GetSdf();
		c[1] = v.rgb;
		if (sdf[1] == 1.0 || isnan(sdf[1]))
			return false;

		localPos = pos + make_int3(1, 1, 0);
		p[2] = make_float3(localPos);
		v = map.FindVoxel(localPos);
		sdf[2] = v.GetSdf();
		c[2] = v.rgb;
		if (sdf[2] == 1.0f || isnan(sdf[2]))
			return false;

		localPos = pos + make_int3(0, 1, 0);
		p[3] = make_float3(localPos);
		v = map.FindVoxel(localPos);
		sdf[3] = v.GetSdf();
		c[3] = v.rgb;
		if (sdf[3] == 1.0f || isnan(sdf[3]))
			return false;

		localPos = pos + make_int3(0, 0, 1);
		p[4] = make_float3(localPos);
		v = map.FindVoxel(localPos);
		sdf[4] = v.GetSdf();
		c[4] = v.rgb;
		if (sdf[4] == 1.0f || isnan(sdf[4]))
			return false;

		localPos = pos + make_int3(1, 0, 1);
		p[5] = make_float3(localPos);
		v = map.FindVoxel(localPos);
		sdf[5] = v.GetSdf();
		c[5] = v.rgb;
		if (sdf[5] == 1.0f || isnan(sdf[5]))
			return false;

		localPos = pos + make_int3(1, 1, 1);
		p[6] = make_float3(localPos);
		v = map.FindVoxel(localPos);
		sdf[6] = v.GetSdf();
		c[6] = v.rgb;
		if (sdf[6] == 1.0f || isnan(sdf[6]))
			return false;

		localPos = pos + make_int3(0, 1, 1);
		p[7] = make_float3(localPos);
		v = map.FindVoxel(localPos);
		sdf[7] = v.GetSdf();
		c[7] = v.rgb;
		if (sdf[7] == 1.0f || isnan(sdf[7]))
			return false;

		if(!FindNormal(n, sdf, pos))
			return false;

		return true;
	}

	template<typename T>
	DEV_FUNC T Interp(T& p1, T& p2, float val1, float val2) {
		if(fabs(0.0f - val1) < 1e-5)
			return p1;
		if(fabs(0.0f - val2) < 1e-5)
			return p2;
		if(fabs(val1 - val2) < 1e-5)
			return p1;
		return p1 + ((0.0f - val1) / (val2 - val1)) * (p2 - p1);
	}

	DEV_FUNC int BuildVertex(float3* vertList, float3* nlist, uchar3* clist, int3 blockPos, int3 localPos)	{
		float3 points[8];
		float3 normal[8];
		uchar3 color[8];
		float sdf[8];

		if (!FindPoint(points, normal, color, sdf, blockPos + localPos))
			return -1;

		int cubeIndex = 0;
		if (sdf[0] < 0)
			cubeIndex |= 1;
		if (sdf[1] < 0)
			cubeIndex |= 2;
		if (sdf[2] < 0)
			cubeIndex |= 4;
		if (sdf[3] < 0)
			cubeIndex |= 8;
		if (sdf[4] < 0)
			cubeIndex |= 16;
		if (sdf[5] < 0)
			cubeIndex |= 32;
		if (sdf[6] < 0)
			cubeIndex |= 64;
		if (sdf[7] < 0)
			cubeIndex |= 128;

		if (edgeTable[cubeIndex] == 0)
			return -1;

		if (edgeTable[cubeIndex] & 1) {
			vertList[0] = Interp<float3>(points[0], points[1], sdf[0], sdf[1]);
//			nlist[0] = Interp<float3>(normal[0], normal[1], sdf[0], sdf[1]);
			clist[0] = Interp<uchar3>(color[0], color[1], sdf[0], sdf[1]);
		}
		if (edgeTable[cubeIndex] & 2) {
			vertList[1] = Interp<float3>(points[1], points[2], sdf[1], sdf[2]);
//			nlist[1] = Interp<float3>(normal[1], normal[2], sdf[1], sdf[2]);
			clist[1] = Interp<uchar3>(color[1], color[2], sdf[1], sdf[2]);
		}
		if (edgeTable[cubeIndex] & 4) {
			vertList[2] = Interp<float3>(points[2], points[3], sdf[2], sdf[3]);
//			nlist[2] = Interp<float3>(normal[2], normal[3], sdf[2], sdf[3]);
			clist[2] = Interp<uchar3>(color[2], color[3], sdf[2], sdf[3]);
		}
		if (edgeTable[cubeIndex] & 8) {
			vertList[3] = Interp<float3>(points[3], points[0], sdf[3], sdf[0]);
//			nlist[3] = Interp<float3>(normal[3], normal[0], sdf[3], sdf[0]);
			clist[3] = Interp<uchar3>(color[3], color[0], sdf[3], sdf[0]);
		}
		if (edgeTable[cubeIndex] & 16) {
			vertList[4] = Interp<float3>(points[4], points[5], sdf[4], sdf[5]);
//			nlist[4] = Interp<float3>(normal[4], normal[5], sdf[4], sdf[5]);
			clist[4] = Interp<uchar3>(color[4], color[5], sdf[4], sdf[5]);
		}
		if (edgeTable[cubeIndex] & 32) {
			vertList[5] = Interp<float3>(points[5], points[6], sdf[5], sdf[6]);
//			nlist[5] = Interp<float3>(normal[5], normal[6], sdf[5], sdf[6]);
			clist[5] = Interp<uchar3>(color[5], color[6], sdf[5], sdf[6]);
		}
		if (edgeTable[cubeIndex] & 64) {
			vertList[6] = Interp<float3>(points[6], points[7], sdf[6], sdf[7]);
//			nlist[6] = Interp<float3>(normal[6], normal[7], sdf[6], sdf[7]);
			clist[6] = Interp<uchar3>(color[6], color[7], sdf[6], sdf[7]);
		}
		if (edgeTable[cubeIndex] & 128) {
			vertList[7] = Interp<float3>(points[7], points[4], sdf[7], sdf[4]);
//			nlist[7] = Interp<float3>(normal[7], normal[4], sdf[7], sdf[4]);
			clist[7] = Interp<uchar3>(color[7], color[4], sdf[7], sdf[4]);
		}
		if (edgeTable[cubeIndex] & 256) {
			vertList[8] = Interp<float3>(points[0], points[4], sdf[0], sdf[4]);
//			nlist[8] = Interp<float3>(normal[0], normal[4], sdf[0], sdf[4]);
			clist[8] = Interp<uchar3>(color[0], color[4], sdf[0], sdf[4]);
		}
		if (edgeTable[cubeIndex] & 512) {
			vertList[9] = Interp<float3>(points[1], points[5], sdf[1], sdf[5]);
//			nlist[9] = Interp<float3>(normal[1], normal[5], sdf[1], sdf[5]);
			clist[9] = Interp<uchar3>(color[1], color[5], sdf[1], sdf[5]);
		}
		if (edgeTable[cubeIndex] & 1024) {
			vertList[10] = Interp<float3>(points[2], points[6], sdf[2], sdf[6]);
//			nlist[10] = Interp<float3>(normal[2], normal[6], sdf[2], sdf[6]);
			clist[10] = Interp<uchar3>(color[2], color[6], sdf[2], sdf[6]);
		}
		if (edgeTable[cubeIndex] & 2048) {
			vertList[11] = Interp<float3>(points[3], points[7], sdf[3], sdf[7]);
//			nlist[11] = Interp<float3>(normal[3], normal[7], sdf[3], sdf[7]);
			clist[11] = Interp<uchar3>(color[3], color[7], sdf[3], sdf[7]);
		}

		return cubeIndex;
	}

	DEV_FUNC void MarchingCube() {
		int x = blockIdx.y * gridDim.x + blockIdx.x;
		if(*noTriangles >= DeviceMap::MaxTriangles)
			return;

		if(x < DeviceMap::NumSdfBlocks && x < *noBlocks) {
			float3 vlist[12];
			float3 nlist[12];
			uchar3 clist[12];
			int3 blockPos = vPos[x] * DeviceMap::BlockSize;
			int3 localPos = make_int3(threadIdx.x, threadIdx.y, threadIdx.z);
			int cubeIdx = BuildVertex(vlist, nlist, clist, blockPos, localPos);
			if(cubeIdx < 0)
				return;
			for(int i = 0; triangleTable.ptr(cubeIdx)[i] != -1; i += 3) {
				int tid = atomicAdd(noTriangles, 1);
				if(tid < DeviceMap::MaxTriangles) {
					triangles[tid * 3 + 0] = vlist[triangleTable.ptr(cubeIdx)[i + 0]] * DeviceMap::VoxelSize;
					triangles[tid * 3 + 1] = vlist[triangleTable.ptr(cubeIdx)[i + 1]] * DeviceMap::VoxelSize;
					triangles[tid * 3 + 2] = vlist[triangleTable.ptr(cubeIdx)[i + 2]] * DeviceMap::VoxelSize;
//				    normals[tid * 3 + 0] = normalised(nlist[triangleTable.ptr(cubeIdx)[i + 0]]);
//					normals[tid * 3 + 1] = normalised(nlist[triangleTable.ptr(cubeIdx)[i + 1]]);
//					normals[tid * 3 + 2] = normalised(nlist[triangleTable.ptr(cubeIdx)[i + 2]]);
					color[tid * 3 + 0] = clist[triangleTable.ptr(cubeIdx)[i + 0]];
					color[tid * 3 + 1] = clist[triangleTable.ptr(cubeIdx)[i + 1]];
					color[tid * 3 + 2] = clist[triangleTable.ptr(cubeIdx)[i + 2]];
				}
			}
		}
	}
};

__global__ void FindExistingBlocks(HashMarchingCube hmc) {
	hmc.FindExistingBlocks();
}

__global__ void MarchingCube(HashMarchingCube hmc) {
	hmc.MarchingCube();
}

uint Mapping::MeshScene() {

	DeviceArray<uint> nBlocks(1);
	DeviceArray<uint> nTriangles(1);
	DeviceArray<int3> nPos(DeviceMap::NumEntries);
	nBlocks.zero();
	nTriangles.zero();

	HashMarchingCube hmc;
	hmc.map = *this;
	hmc.triangleTable = mTriTable;
	hmc.edgeTable = mEdgeTable;
	hmc.triangles = mMesh;
	hmc.noBlocks = nBlocks;
	hmc.normals = mMeshNormal;
	hmc.noTriangles = nTriangles;
	hmc.color = mColorMap;
	hmc.vPos = nPos;

	dim3 block(MaxThread);
	dim3 grid(cv::divUp((int)DeviceMap::NumEntries, block.x));
	FindExistingBlocks<<<grid, block>>>(hmc);
	SafeCall(hipGetLastError());
	SafeCall(hipDeviceSynchronize());

	uint n;
	nBlocks.download((void*)&n);
//	std::cout << n << std::endl;
	if(n == 0)
		return 0;

	block = dim3(8, 8, 8);
	grid = dim3(cv::divUp((int)n, 16), 16);
	MarchingCube<<<grid, block>>>(hmc);
	SafeCall(hipGetLastError());
	SafeCall(hipDeviceSynchronize());

	nTriangles.download((void*)&n);
	nTriangle = min(n, DeviceMap::MaxTriangles);

	mMutexMesh.lock();
	bUpdated = true;
	mMutexMesh.unlock();
	return nTriangle;
}
