#include "hip/hip_runtime.h"
#include "Mapping.hpp"

template<int threadBlock>
DEV_FUNC int ComputeOffset(uint element, uint *sum) {

	__shared__ uint buffer[threadBlock];
	__shared__ uint blockOffset;

	if (threadIdx.x == 0)
		memset(buffer, 0, sizeof(uint) * 16 * 16);
	__syncthreads();

	buffer[threadIdx.x] = element;
	__syncthreads();

	int s1, s2;

	for (s1 = 1, s2 = 1; s1 < threadBlock; s1 <<= 1) {
		s2 |= s1;
		if ((threadIdx.x & s2) == s2)
			buffer[threadIdx.x] += buffer[threadIdx.x - s1];
		__syncthreads();
	}

	for (s1 >>= 2, s2 >>= 1; s1 >= 1; s1 >>= 1, s2 >>= 1) {
		if (threadIdx.x != threadBlock - 1 && (threadIdx.x & s2) == s2)
			buffer[threadIdx.x + s1] += buffer[threadIdx.x];
		__syncthreads();
	}

	if (threadIdx.x == 0 && buffer[threadBlock - 1] > 0)
		blockOffset = atomicAdd(sum, buffer[threadBlock - 1]);
	__syncthreads();

	int offset;
	if (threadIdx.x == 0) {
		if (buffer[threadIdx.x] == 0)
			offset = -1;
		else
			offset = blockOffset;
	} else {
		if (buffer[threadIdx.x] == buffer[threadIdx.x - 1])
			offset = -1;
		else
			offset = blockOffset + buffer[threadIdx.x - 1];
	}

	return offset;
}

struct HashMarchingCube {

	DeviceMap map;
	PtrSz<float3> triangles;
	PtrStep<int> triangleTable;
	PtrSz<int> edgeTable;
	PtrSz<int3> vPos;
	uint* noBlocks;
	uint* noTriangles;
	Matrix3f Rot, invRot;
	float3 trans;
	int cols, rows;
	float fx, fy, cx, cy;

	DEV_FUNC void FindExistingBlocks() {
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		__shared__ bool scan;
		if(x == 0)
			scan = false;
		__syncthreads();
		uint val = 0;
		if (x < DeviceMap::NumEntries) {
			if (map.hashEntries[x].ptr >= 0) {
				scan = true;
				val = 1;
			}
		}
		__syncthreads();
		if(scan) {
			int offset = ComputeOffset<1024>(val, noBlocks);
			if(offset != -1) {
				vPos[offset] = map.hashEntries[x].pos;
			}
		}
//		if (x < DeviceMap::NumEntries) {
//			if(map.hashEntries[x].ptr >= 0) {
//				uint offset = atomicAdd(noBlocks, 1);
//				vPos[offset] = map.hashEntries[x].pos;
//			}
//		}
	}

	DEV_FUNC bool FindPoint(float3* p, float* sdf, int3 pos) {

		int3 localPos;

		localPos = pos + make_int3(0, 0, 0);
		p[0] = make_float3(localPos);
		sdf[0] = map.FindVoxel(localPos).GetSdf();
		if (sdf[0] == 1.0f || isnan(sdf[0]) || sdf[0] < -0.5)
			return false;

		localPos = pos + make_int3(1, 0, 0);
		p[1] = make_float3(localPos);
		sdf[1] = map.FindVoxel(localPos).GetSdf();
		if (sdf[1] == 1.0f || isnan(sdf[1]) || sdf[1] < -0.5)
			return false;

		localPos = pos + make_int3(1, 1, 0);
		p[2] = make_float3(localPos);
		sdf[2] = map.FindVoxel(localPos).GetSdf();
		if (sdf[2] == 1.0f || isnan(sdf[2]) || sdf[2] < -0.5)
			return false;

		localPos = pos + make_int3(0, 1, 0);
		p[3] = make_float3(localPos);
		sdf[3] = map.FindVoxel(localPos).GetSdf();
		if (sdf[3] == 1.0f || isnan(sdf[3]) || sdf[3] < -0.5)
			return false;

		localPos = pos + make_int3(0, 0, 1);
		p[4] = make_float3(localPos);
		sdf[4] = map.FindVoxel(localPos).GetSdf();
		if (sdf[4] == 1.0f || isnan(sdf[4]) || sdf[4] <= -0.5)
			return false;

		localPos = pos + make_int3(1, 0, 1);
		p[5] = make_float3(localPos);
		sdf[5] = map.FindVoxel(localPos).GetSdf();
		if (sdf[5] == 1.0f || isnan(sdf[5]) || sdf[5] <= -0.5)
			return false;

		localPos = pos + make_int3(1, 1, 1);
		p[6] = make_float3(localPos);
		sdf[6] = map.FindVoxel(localPos).GetSdf();
		if (sdf[6] == 1.0f || isnan(sdf[6]) || sdf[6] <= -0.5)
			return false;

		localPos = pos + make_int3(0, 1, 1);
		p[7] = make_float3(localPos);
		sdf[7] = map.FindVoxel(localPos).GetSdf();
		if (sdf[7] == 1.0f || isnan(sdf[7]) || sdf[7] <= -0.5)
			return false;

		return true;
	}

	DEV_FUNC float3 Interp(float3& p1, float3& p2, float val1, float val2) {
		if(fabs(0.0f - val1) < 1e-5)
			return p1;
		if(fabs(0.0f - val2) < 1e-5)
			return p2;
		if(fabs(val1 - val2) < 1e-5)
			return p1;
		return p1 + ((0.0f - val1) / (val2 - val1)) * (p2 - p1);
	}

	DEV_FUNC int BuildVertex(float3* vertList, int3 blockPos, int3 localPos)
		{
		float3 points[8];
		float sdf[8];

		if (!FindPoint(points, sdf, blockPos + localPos))
			return -1;

		int cubeIndex = 0;
		if (sdf[0] < 0)
			cubeIndex |= 1;
		if (sdf[1] < 0)
			cubeIndex |= 2;
		if (sdf[2] < 0)
			cubeIndex |= 4;
		if (sdf[3] < 0)
			cubeIndex |= 8;
		if (sdf[4] < 0)
			cubeIndex |= 16;
		if (sdf[5] < 0)
			cubeIndex |= 32;
		if (sdf[6] < 0)
			cubeIndex |= 64;
		if (sdf[7] < 0)
			cubeIndex |= 128;

		if (edgeTable[cubeIndex] == 0)
			return -1;

		if (edgeTable[cubeIndex] & 1)
			vertList[0] = Interp(points[0], points[1], sdf[0], sdf[1]);
		if (edgeTable[cubeIndex] & 2)
			vertList[1] = Interp(points[1], points[2], sdf[1], sdf[2]);
		if (edgeTable[cubeIndex] & 4)
			vertList[2] = Interp(points[2], points[3], sdf[2], sdf[3]);
		if (edgeTable[cubeIndex] & 8)
			vertList[3] = Interp(points[3], points[0], sdf[3], sdf[0]);
		if (edgeTable[cubeIndex] & 16)
			vertList[4] = Interp(points[4], points[5], sdf[4], sdf[5]);
		if (edgeTable[cubeIndex] & 32)
			vertList[5] = Interp(points[5], points[6], sdf[5], sdf[6]);
		if (edgeTable[cubeIndex] & 64)
			vertList[6] = Interp(points[6], points[7], sdf[6], sdf[7]);
		if (edgeTable[cubeIndex] & 128)
			vertList[7] = Interp(points[7], points[4], sdf[7], sdf[4]);
		if (edgeTable[cubeIndex] & 256)
			vertList[8] = Interp(points[0], points[4], sdf[0], sdf[4]);
		if (edgeTable[cubeIndex] & 512)
			vertList[9] = Interp(points[1], points[5], sdf[1], sdf[5]);
		if (edgeTable[cubeIndex] & 1024)
			vertList[10] = Interp(points[2], points[6], sdf[2], sdf[6]);
		if (edgeTable[cubeIndex] & 2048)
			vertList[11] = Interp(points[3], points[7], sdf[3], sdf[7]);

		return cubeIndex;
	}

	DEV_FUNC void MarchingCube() {
		int x = blockIdx.y * gridDim.x + blockIdx.x;
		if(*noTriangles >= DeviceMap::MaxTriangles)
			return;

		if(x < DeviceMap::NumSdfBlocks && x < *noBlocks) {
			float3 vlist[12];
			int3 blockPos = vPos[x] * DeviceMap::BlockSize;
			int3 localPos = map.localIdxToLocalPos(threadIdx.x);
			int cubeIdx = BuildVertex(vlist, blockPos, localPos);
			if(cubeIdx < 0)
				return;
			for(int i = 0; triangleTable.ptr(cubeIdx)[i] != -1; i += 3) {

				int tid = atomicAdd(noTriangles, 1);
				if(tid < DeviceMap::MaxTriangles) {
					triangles[tid * 3 + 0] = vlist[triangleTable.ptr(cubeIdx)[i + 0]] * DeviceMap::VoxelSize;
					triangles[tid * 3 + 1] = vlist[triangleTable.ptr(cubeIdx)[i + 1]] * DeviceMap::VoxelSize;
					triangles[tid * 3 + 2] = vlist[triangleTable.ptr(cubeIdx)[i + 2]] * DeviceMap::VoxelSize;
					if(isnan(triangles[tid*3 + 0].x))
						printf("%f\n", triangles[tid*3+0].x);
				}
			}
		}
	}
};

__global__ void FindExistingBlocks(HashMarchingCube hmc) {
	hmc.FindExistingBlocks();
}

__global__ void MarchingCube(HashMarchingCube hmc) {
	hmc.MarchingCube();
}

uint Mapping::MeshScene() {

	DeviceArray<uint> nBlocks(1);
	DeviceArray<uint> nTriangles(1);
	DeviceArray<int3> nPos(DeviceMap::NumEntries);
	nBlocks.zero();
	nTriangles.zero();

	HashMarchingCube mesh;
	mesh.map = *this;
	mesh.triangleTable = mTriTable;
	mesh.edgeTable = mEdgeTable;
	mesh.triangles = mMesh;
	mesh.noBlocks = nBlocks;
	mesh.noTriangles = nTriangles;
	mesh.vPos = nPos;

	dim3 block(MaxThread);
	dim3 grid(cv::divUp((int)DeviceMap::NumEntries, block.x));
	FindExistingBlocks<<<grid, block>>>(mesh);
	SafeCall(hipGetLastError());
	SafeCall(hipDeviceSynchronize());

	uint n;
	nBlocks.download((void*)&n);
	std::cout << n << std::endl;

	block = dim3(512);
	grid = dim3(cv::divUp((int)n, 16), 16);
	MarchingCube<<<grid, block>>>(mesh);
	SafeCall(hipGetLastError());
	SafeCall(hipDeviceSynchronize());

	nTriangles.download((void*)&n);
	mMesh.download((void*)mHostMesh);
	nTriangle = min(n, DeviceMap::MaxTriangles);
	return nTriangle;
}
