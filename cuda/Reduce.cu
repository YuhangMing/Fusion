#include "hip/hip_runtime.h"
#include "Frame.h"
#include "Converter.h"
#include "DeviceMath.h"
#include "DeviceFunc.h"
#include "DeviceArray.h"

#define WarpSize 32
#define MaxThread 1024
#define ICPWeight 5

template<typename T, int size> __device__
inline void WarpReduceSum(T* val) {
#pragma unroll
	for(int offset = WarpSize / 2; offset > 0; offset /= 2) {
#pragma unroll
		for(int i = 0; i < size; ++i) {
			val[i] += __shfl_down_sync(0xffffffff, val[i], offset);
		}
	}
}

template<typename T, int size> __device__
inline void BlockReduceSum(T* val) {
	static __shared__ T shared[32 * size];
	int lane = threadIdx.x % warpSize;
	int wid = threadIdx.x / warpSize;

	WarpReduceSum<T, size>(val);

    if(lane == 0)
    	memcpy(&shared[wid * size], val, sizeof(T) * size);

    __syncthreads();

    if(threadIdx.x < blockDim.x / warpSize)
    	memcpy(val, &shared[lane * size], sizeof(T) * size);
    else
    	memset(val, 0, sizeof(T) * size);

    if(wid == 0)
        WarpReduceSum<T, size>(val);
}

template<typename T, int size> __global__
void ReduceSum(PtrStep<T> in, T* out, int N) {
	T sum[size];
	memset(sum, 0, sizeof(T) * size);
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    for(; i < N; i += blockDim.x * gridDim.x)
#pragma unroll
		for(int j = 0; j < size; ++j)
			sum[j] += in.ptr(i)[j];

    BlockReduceSum<T, size>(sum);

    if(threadIdx.x == 0)
#pragma unroll
		for(int i = 0; i < size; ++i)
			out[i] = sum[i];
}

struct ICPReduce {

	bool ICPOnly;
	Matrix3f Rcurr;
	Matrix3f Rlast;
	Matrix3f invRlast;
	Matrix3f R;
	float3 t;
	float3 tcurr;
	float3 tlast;
	PtrStep<float4> VMapCurr, VMapLast;
	PtrStep<float3> NMapCurr, NMapLast;
	PtrStep<float> dIx, dIy;
	PtrStep<uchar> GrayCurr, GrayLast;
	int cols, rows, N;
	float fx, fy, cx, cy;
	float angleThresh, distThresh;
	mutable PtrStepSz<float> out;

	__device__ inline
	bool SearchCorresp(int& x, int& y, int& u, int& v, float3& vcurr_g,
					   float3& vlast_g, float3& nlast_g, float3& vcurr_p) const {

		float3 vcurr_c = make_float3(VMapCurr.ptr(y)[x]);
		if(isnan(vcurr_c.x) || vcurr_c.z < 1e-3)
			return false;

		vcurr_g = Rcurr * vcurr_c + tcurr;
		vcurr_p = invRlast * (vcurr_g - tlast);

		float invz = 1.0 / vcurr_p.z;
		u = __float2int_rd(vcurr_p.x * invz * fx + cx + 0.5);
		v = __float2int_rd(vcurr_p.y * invz * fy + cy + 0.5);
		if(u < 0 || v < 0 || u >= cols || v >= rows)
			return false;

		float3 vlast_c = make_float3(VMapLast.ptr(v)[u]);
		vlast_g = Rlast * vlast_c + tlast;

		float3 ncurr_c = NMapCurr.ptr(y)[x];
		float3 ncurr_g = Rcurr * ncurr_c;

		float3 nlast_c = NMapLast.ptr(v)[u];
		nlast_g = Rlast * nlast_c;

		float dist = norm(vlast_g - vcurr_g);
		float sine = norm(cross(ncurr_g, nlast_g));

		return (sine < angleThresh && dist <= distThresh &&
				!isnan(ncurr_c.x) && !isnan(nlast_c.x));
	}

	__device__ inline
	bool ComputeRGB(int& x, int& y, int& u, int& v,
					float3& vcg, float3& vcp, float* row) const {

		float dx = dIx.ptr(v)[u];
		float dy = dIy.ptr(v)[u];
		if((dx == 0 || dy == 0) || vcp.z < 1e-2 || vcp.z > 3.0)
			return false;

		float3 rcx = -invRlast.coloumx();
		float3 rcy = -invRlast.coloumy();
		float3 rcz = -invRlast.coloumz();
		float3 dIdh;
		dIdh.x = dx * fx / vcp.z;
		dIdh.y = dy * fy / vcp.z;
		dIdh.z = -(dx * fx * vcp.x  + dy * fy * vcp.y) / (vcp.z * vcp.z);
		float3 r0xp = cross(invRlast.rowx, vcg);
		float3 r1xp = cross(invRlast.rowy, vcg);
		float3 r2xp = cross(invRlast.rowz, vcg);

		row[0] = dIdh * rcx;
		row[1] = dIdh * rcy;
		row[2] = dIdh * rcz;
		row[3] = dIdh * make_float3(r0xp.x, r1xp.x, r2xp.x);
		row[4] = dIdh * make_float3(r0xp.y, r1xp.y, r2xp.y);
		row[5] = dIdh * make_float3(r0xp.z, r1xp.z, r2xp.z);
		row[6] = -(GrayCurr.ptr(y)[x] - GrayLast.ptr(v)[u]);
		return true;
	}

	template<bool bUseRGB> __device__ inline
	void GetRow(int& i, float* sum) const {
		int y = i / cols;
		int x = i - (y * cols);

		int u = 0, v = 0;
		bool bCorresp = false, bRGB = false;
		float3 vcurr, vlast, nlast, vcurrp;
		bCorresp = SearchCorresp(x, y, u, v, vcurr, vlast, nlast, vcurrp);
		float row[7] = { 0, 0, 0, 0, 0, 0, 0 };
		float row_rgb[7] = { 0, 0, 0, 0, 0, 0, 0 };

		if(bCorresp) {
			float3 nvcross = cross(nlast, vcurr);
            row[0] = -nlast.x;
            row[1] = -nlast.y;
            row[2] = -nlast.z;
            row[3] = nvcross.x;
            row[4] = nvcross.y;
            row[5] = nvcross.z;
            row[6] = -nlast * (vlast - vcurr);

            if(bUseRGB) {
            	bRGB = ComputeRGB(x, y, u, v, vcurr, vcurrp, row_rgb);
            }
		}

		int count = 0;
		if(!bUseRGB || !bRGB) {
#pragma unroll
			for(int i = 0; i < 7; ++i)
#pragma unroll
				for(int j = i; j < 7; ++j) {
					sum[count++] = row[i] * row[j];
				}
		}
		else {
#pragma unroll
			for(int i = 0; i < 7; ++i)
#pragma unroll
				for(int j = i; j < 7; ++j) {
					sum[count++] = 0.1 * (row[i] * row[j]) + 0.9 * (row_rgb[i] * row_rgb[j]);
//					sum[count++] = row_rgb[i] * row_rgb[j];
//					sum[count++] = row[i] * row[j];
				}
		}
		sum[count] = (float)bCorresp;
	}

	template<typename T, int size, bool bRGB>
	__device__ void operator()() const {
		T sum[size];
		T val[size];
		memset(sum, 0, sizeof(T) * size);
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		for (; i < N; i += blockDim.x * gridDim.x) {
			memset(val, 0, sizeof(T) * size);
			GetRow<bRGB>(i, val);

#pragma unroll
			for(int j = 0; j < size; ++j)
				sum[j] += val[j];
		}

		BlockReduceSum<T, size>(sum);

		if (threadIdx.x == 0)
#pragma unroll
			for(int i = 0; i < size; ++i)
				out.ptr(blockIdx.x)[i] = sum[i];
	}
};

__global__ void
ICPReduceSum_device(const ICPReduce icp) {
	icp.template operator()<float, 29, true>();
}

static void inline
CreateMatrix(float* host_data, float* host_a, float* host_b) {
    int shift = 0;
	for (int i = 0; i < 6; ++i)
		for (int j = i; j < 7; ++j) {
			float value = host_data[shift++];
			if (j == 6)
				host_b[i] = value;
			else
				host_a[j * 6 + i] = host_a[i * 6 + j] = value;
		}
}

void ICPReduceSum(Frame& NextFrame, Frame& LastFrame, int pyrnum,
				  float* host_a, float* host_b, float& cost) {

	DeviceArray2D<float> sum(29, 96);
	DeviceArray<float> result(29);
	result.zero();
	sum.zero();

	ICPReduce icp;
	icp.out = sum;
	icp.dIx = LastFrame.mdIx[pyrnum];
	icp.dIy = LastFrame.mdIy[pyrnum];
	icp.VMapCurr = NextFrame.mVMap[pyrnum];
	icp.NMapCurr = NextFrame.mNMap[pyrnum];
	icp.GrayCurr = NextFrame.mGray[pyrnum];
	icp.VMapLast = LastFrame.mVMap[pyrnum];
	icp.NMapLast = LastFrame.mNMap[pyrnum];
	icp.GrayLast = LastFrame.mGray[pyrnum];
	icp.cols = Frame::cols(pyrnum);
	icp.rows = Frame::rows(pyrnum);
	icp.N = Frame::pixels(pyrnum);
	icp.Rcurr = NextFrame.mRcw;
	icp.tcurr = Converter::CvMatToFloat3(NextFrame.mtcw);
	icp.Rlast = LastFrame.mRcw;
	icp.invRlast = LastFrame.mRwc;
	icp.tlast = Converter::CvMatToFloat3(LastFrame.mtcw);
	icp.angleThresh = 0.6;
	icp.distThresh = 0.1;
	icp.ICPOnly = false;
	icp.fx = Frame::fx(pyrnum);
	icp.fy = Frame::fy(pyrnum);
	icp.cx = Frame::cx(pyrnum);
	icp.cy = Frame::cy(pyrnum);

	ICPReduceSum_device<<<96, 224>>>(icp);
	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());

	ReduceSum<float, 29><<<1, MaxThread>>>(sum, result, 96);
	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());

	float host_data[29];
	result.download(host_data);
	CreateMatrix(host_data, host_a, host_b);
	cost = sqrt(host_data[27]) / host_data[28];
}

//struct ICPReduce {
//	Matrix3f Rcurr;
//	Matrix3f Rlast;
//	Matrix3f invRlast;
//	Matrix3f R;
//	float3 t;
//	float3 tcurr;
//	float3 tlast;
//	PtrStep<float4> VMapCurr, VMapLast;
//	PtrStep<float3> NMapCurr, NMapLast;
//	int cols, rows, N;
//	float fx, fy, cx, cy;
//	float angleThresh, distThresh;
//	mutable PtrStepSz<float> out;
//
//	__device__ inline
//	bool SearchCorresp(int& x, int& y, float3& vcurr_g, float3& vlast_g, float3& nlast_g) const {
//
//		float3 vcurr_c = make_float3(VMapCurr.ptr(y)[x]);
//		if(isnan(vcurr_c.x) || vcurr_c.z < 1e-3)
//			return false;
//
//		vcurr_g = Rcurr * vcurr_c + tcurr;
//		float3 vcurr_p = invRlast * (vcurr_g - tlast);
//
//		float invz = 1.0 / vcurr_p.z;
//		int u = __float2int_rd(vcurr_p.x * invz * fx + cx + 0.5);
//		int v = __float2int_rd(vcurr_p.y * invz * fy + cy + 0.5);
//		if(u < 0 || v < 0 || u >= cols || v >= rows)
//			return false;
//
//		float3 vlast_c = make_float3(VMapLast.ptr(v)[u]);
//		vlast_g = Rlast * vlast_c + tlast;
//
//		float3 ncurr_c = NMapCurr.ptr(y)[x];
//		float3 ncurr_g = Rcurr * ncurr_c;
//
//		float3 nlast_c = NMapLast.ptr(v)[u];
//		nlast_g = Rlast * nlast_c;
//
//		float dist = norm(vlast_g - vcurr_g);
//		float sine = norm(cross(ncurr_g, nlast_g));
//
//		return (sine < angleThresh && dist <= distThresh && !isnan(ncurr_c.x) && !isnan(nlast_c.x));
//	}
//
//	__device__ inline
//	void GetRow(int& i, float* sum) const {
//		int y = i / cols;
//		int x = i - (y * cols);
//
//		bool bCorresp = false;
//		float3 vcurr, vlast, nlast;
//		bCorresp = SearchCorresp(x, y, vcurr, vlast, nlast);
//		float row[7] = { 0, 0, 0, 0, 0, 0, 0 };
//
//		if(bCorresp) {
//			float3 nvcross = cross(nlast, vcurr);
//            row[0] = -nlast.x;
//            row[1] = -nlast.y;
//            row[2] = -nlast.z;
//            row[3] = nvcross.x;
//            row[4] = nvcross.y;
//            row[5] = nvcross.z;
//            row[6] = (vcurr - vlast) * nlast;
//		}
//
//		int count = 0;
//#pragma unroll
//		for(int i = 0; i < 7; ++i)
//#pragma unroll
//			for(int j = i; j < 7; ++j) {
//				sum[count++] = row[i] * row[j];
//			}
//		sum[count] = (float)bCorresp;
//	}
//
//	template<typename T, int size>
//	__device__ void operator()() const {
//		T sum[size];
//		T val[size];
//		memset(sum, 0, sizeof(T) * size);
//		int i = blockIdx.x * blockDim.x + threadIdx.x;
//		for (; i < N; i += blockDim.x * gridDim.x) {
//			memset(val, 0, sizeof(T) * size);
//			GetRow(i, val);
//
//#pragma unroll
//			for(int j = 0; j < size; ++j)
//				sum[j] += val[j];
//		}
//
//		BlockReduceSum<T, size>(sum);
//
//		if (threadIdx.x == 0)
//#pragma unroll
//			for(int i = 0; i < size; ++i)
//				out.ptr(blockIdx.x)[i] = sum[i];
//	}
//};
//
//__global__ void
//ICPReduceSum_device(const ICPReduce icp) {
//	icp.template operator()<float, 29>();
//}
//
//static void inline
//CreateMatrix(float* host_data, float* host_a, float* host_b) {
//    int shift = 0;
//	for (int i = 0; i < 6; ++i)
//		for (int j = i; j < 7; ++j) {
//			float value = host_data[shift++];
//			if (j == 6)
//				host_b[i] = value;
//			else
//				host_a[j * 6 + i] = host_a[i * 6 + j] = value;
//		}
//}
//
//void ICPReduceSum(Frame& NextFrame, Frame& LastFrame, int PyrLevel, float* host_a, float* host_b, float& cost) {
//
//	DeviceArray2D<float> sum(29, 96);
//	DeviceArray<float> result(29);
//	result.zero();
//	sum.zero();
//
//	ICPReduce icp;
//	icp.out = sum;
//	icp.VMapCurr = NextFrame.mVMap[PyrLevel];
//	icp.NMapCurr = NextFrame.mNMap[PyrLevel];
//	icp.VMapLast = LastFrame.mVMap[PyrLevel];
//	icp.NMapLast = LastFrame.mNMap[PyrLevel];
//	icp.cols = Frame::cols(PyrLevel);
//	icp.rows = Frame::rows(PyrLevel);
//	icp.N = Frame::pixels(PyrLevel);
//	icp.Rcurr = NextFrame.mRcw;
//	icp.tcurr = Converter::CvMatToFloat3(NextFrame.mtcw);
//	icp.Rlast = LastFrame.mRcw;
//	icp.invRlast = LastFrame.mRwc;
//	icp.tlast = Converter::CvMatToFloat3(LastFrame.mtcw);
//	icp.angleThresh = 0.6;
//	icp.distThresh = 0.1;
//	icp.fx = Frame::fx(PyrLevel);
//	icp.fy = Frame::fy(PyrLevel);
//	icp.cx = Frame::cx(PyrLevel);
//	icp.cy = Frame::cy(PyrLevel);
//
//	ICPReduceSum_device<<<96, 224>>>(icp);
//	SafeCall(hipDeviceSynchronize());
//	SafeCall(hipGetLastError());
//
//	ReduceSum<float, 29><<<1, MaxThread>>>(sum, result, 96);
//	SafeCall(hipDeviceSynchronize());
//	SafeCall(hipGetLastError());
//
//	float host_data[29];
//	result.download(host_data);
//	CreateMatrix(host_data, host_a, host_b);
//	cost = sqrt(host_data[27]) / host_data[28];
//}
