#include "hip/hip_runtime.h"
#include "Frame.h"
#include "Converter.h"
#include "DeviceMath.h"
#include "DeviceFunc.h"
#include "DeviceArray.h"

#define WarpSize 32
#define MaxThread 1024
#define ICPWeight 5

template<typename T, int size> __device__
inline void WarpReduceSum(T* val) {
#pragma unroll
	for(int offset = WarpSize / 2; offset > 0; offset /= 2) {
#pragma unroll
		for(int i = 0; i < size; ++i) {
			val[i] += __shfl_down_sync(0xffffffff, val[i], offset);
		}
	}
}

template<typename T, int size> __device__
inline void BlockReduceSum(T* val) {
	static __shared__ T shared[32 * size];
	int lane = threadIdx.x % warpSize;
	int wid = threadIdx.x / warpSize;

	WarpReduceSum<T, size>(val);

    if(lane == 0)
    	memcpy(&shared[wid * size], val, sizeof(T) * size);

    __syncthreads();

    if(threadIdx.x < blockDim.x / warpSize)
    	memcpy(val, &shared[lane * size], sizeof(T) * size);
    else
    	memset(val, 0, sizeof(T) * size);

    if(wid == 0)
        WarpReduceSum<T, size>(val);
}

template<typename T, int size> __global__
void ReduceSum(PtrStep<T> in, T* out, int N) {
	T sum[size];
	memset(sum, 0, sizeof(T) * size);
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    for(; i < N; i += blockDim.x * gridDim.x)
#pragma unroll
		for(int j = 0; j < size; ++j)
			sum[j] += in.ptr(i)[j];

    BlockReduceSum<T, size>(sum);

    if(threadIdx.x == 0)
#pragma unroll
		for(int i = 0; i < size; ++i)
			out[i] = sum[i];
}

struct ICPReduce {

	bool ICPOnly;
	float icpW;
	Matrix3f Rcurr;
	Matrix3f Rlast;
	Matrix3f invRlast;
	Matrix3f R;
	float3 t;
	float3 tcurr;
	float3 tlast;
	PtrStep<float> dIx, dIy;
	PtrStep<float4> VMapCurr, VMapLast;
	PtrStep<float3> NMapCurr, NMapLast;
	PtrStep<uchar> GrayCurr, GrayLast;
	int cols, rows, N;
	float fx, fy, cx, cy;
	float angleThresh, distThresh;

	mutable float w;
	mutable PtrStepSz<float> out;

	__device__ inline
	bool SearchCorresp(bool& view, int& x, int& y, int& u, int& v, float3& vcurr_g,
					   float3& vlast_g, float3& nlast_g, float3& vcurr_p) const {

		float3 vcurr_c = make_float3(VMapCurr.ptr(y)[x]);
		if(isnan(vcurr_c.x) || vcurr_c.z < 1e-3)
			return false;

		vcurr_g = Rcurr * vcurr_c + tcurr;
		vcurr_p = invRlast * (vcurr_g - tlast);

		float invz = 1.0 / vcurr_p.z;
		u = __float2int_rd(vcurr_p.x * invz * fx + cx + 0.5);
		v = __float2int_rd(vcurr_p.y * invz * fy + cy + 0.5);
		if(u < 0 || v < 0 || u >= cols || v >= rows)
			return false;

		view = true;
		float3 vlast_c = make_float3(VMapLast.ptr(v)[u]);
		vlast_g = Rlast * vlast_c + tlast;

		float3 ncurr_c = NMapCurr.ptr(y)[x];
		float3 ncurr_g = Rcurr * ncurr_c;

		float3 nlast_c = NMapLast.ptr(v)[u];
		nlast_g = Rlast * nlast_c;

		float dist = norm(vlast_g - vcurr_g);
		float sine = norm(cross(ncurr_g, nlast_g));

		return (sine < angleThresh && dist <= distThresh &&
				!isnan(ncurr_c.x) && !isnan(nlast_c.x));
	}

	__device__ inline
	bool ComputeRGB(int& x, int& y, int& u, int& v,
					float3& vcg, float3& vcp, float* row) const {

		float dx = dIx.ptr(v)[u];
		float dy = dIy.ptr(v)[u];

		bool valid = true;
		const int r = 2;
		for(int i = max(0, u - r ); i < min(u + r + 1, cols); ++i)
			for(int j = max(0, v - r ); j < min(v + r + 1, rows); ++j)
				valid = (GrayLast.ptr(j)[i] > 0) && (GrayCurr.ptr(j)[i] > 0) && valid;

		if(!valid)
			return false;

		float3 rcx = -invRlast.coloumx();
		float3 rcy = -invRlast.coloumy();
		float3 rcz = -invRlast.coloumz();
		float3 dIdh;
		dIdh.x = dx * fx / vcp.z;
		dIdh.y = dy * fy / vcp.z;
		dIdh.z = -(dx * fx * vcp.x  + dy * fy * vcp.y) / (vcp.z * vcp.z);
		float3 r0xp = cross(invRlast.rowx, vcg);
		float3 r1xp = cross(invRlast.rowy, vcg);
		float3 r2xp = cross(invRlast.rowz, vcg);

		row[0] = dIdh * rcx;
		row[1] = dIdh * rcy;
		row[2] = dIdh * rcz;
		row[3] = dIdh * make_float3(r0xp.x, r1xp.x, r2xp.x);
		row[4] = dIdh * make_float3(r0xp.y, r1xp.y, r2xp.y);
		row[5] = dIdh * make_float3(r0xp.z, r1xp.z, r2xp.z);
		row[6] = -(GrayCurr.ptr(y)[x] - GrayLast.ptr(v)[u]);

		return true;
	}

	template<bool bUseRGB> __device__ inline
	void GetRow(int& i, float* sum) const {
		int y = i / cols;
		int x = i - (y * cols);

		int u = 0, v = 0;
		bool bCorresp = false, bRGB = false;
		bool bView = false;
		float3 vcurr, vlast, nlast, vcurrp;
		bCorresp = SearchCorresp(bView, x, y, u, v, vcurr, vlast, nlast, vcurrp);
		float row[7] = { 0, 0, 0, 0, 0, 0, 0 };
		float row_rgb[7] = { 0, 0, 0, 0, 0, 0, 0 };

		if(bCorresp) {
			float3 nvcross = cross(nlast, vcurr);
            row[0] = -nlast.x;
            row[1] = -nlast.y;
            row[2] = -nlast.z;
            row[3] = nvcross.x;
            row[4] = nvcross.y;
            row[5] = nvcross.z;
            row[6] = -nlast * (vlast - vcurr);
		}

        if(bView && bUseRGB) {
        	bRGB = ComputeRGB(x, y, u, v, vcurr, vcurrp, row_rgb);
        }

		int count = 0;
		if(!bUseRGB || !bRGB) {
#pragma unroll
			for(int i = 0; i < 7; ++i)
#pragma unroll
				for(int j = i; j < 7; ++j)
//					sum[count++] = row[i] * row[j];
					sum[count++] = row_rgb[i] * row_rgb[j];
		}
		else {
#pragma unroll
			for(int i = 0; i < 7; ++i)
#pragma unroll
				for(int j = i; j < 7; ++j)
					sum[count++] = icpW * row[i] * row[j] + (1 - icpW) * row_rgb[i] * row_rgb[j];
//					sum[count++] = row_rgb[i] * row_rgb[j];
//					sum[count++] = row[i] * row[j];
		}
		sum[count] = (float)bCorresp;
	}

	template<typename T, int size, bool bRGB>
	__device__ void operator()() const {
		T sum[size];
		T val[size];
		memset(sum, 0, sizeof(T) * size);
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		for (; i < N; i += blockDim.x * gridDim.x) {
			memset(val, 0, sizeof(T) * size);
			GetRow<bRGB>(i, val);

#pragma unroll
			for(int j = 0; j < size; ++j)
				sum[j] += val[j];
		}

		BlockReduceSum<T, size>(sum);

		if (threadIdx.x == 0)
#pragma unroll
			for(int i = 0; i < size; ++i)
				out.ptr(blockIdx.x)[i] = sum[i];
	}
};

__global__ void
ICPReduceSum_device(const ICPReduce icp) {
	icp.template operator()<float, 29, true>();
}

static void inline
CreateMatrix(float* host_data, float* host_a, float* host_b) {
    int shift = 0;
	for (int i = 0; i < 6; ++i)
		for (int j = i; j < 7; ++j) {
			float value = host_data[shift++];
			if (j == 6)
				host_b[i] = value;
			else
				host_a[j * 6 + i] = host_a[i * 6 + j] = value;
		}
}

void ICPReduceSum(Frame& NextFrame, Frame& LastFrame, int pyrnum,
				  float* host_a, float* host_b, float& cost) {

	DeviceArray2D<float> sum(29, 96);
	DeviceArray<float> result(29);
	result.zero();
	sum.zero();

	ICPReduce icp;
	icp.out = sum;
	icp.dIx = LastFrame.mdIx[pyrnum];
	icp.dIy = LastFrame.mdIy[pyrnum];
	icp.VMapCurr = NextFrame.mVMap[pyrnum];
	icp.NMapCurr = NextFrame.mNMap[pyrnum];
	icp.GrayCurr = NextFrame.mGray[pyrnum];
	icp.VMapLast = LastFrame.mVMap[pyrnum];
	icp.NMapLast = LastFrame.mNMap[pyrnum];
	icp.GrayLast = LastFrame.mGray[pyrnum];
	icp.cols = Frame::cols(pyrnum);
	icp.rows = Frame::rows(pyrnum);
	icp.N = Frame::pixels(pyrnum);
	icp.Rcurr = NextFrame.mRcw;
	icp.tcurr = Converter::CvMatToFloat3(NextFrame.mtcw);
	icp.Rlast = LastFrame.mRcw;
	icp.invRlast = LastFrame.mRwc;
	icp.tlast = Converter::CvMatToFloat3(LastFrame.mtcw);
	icp.angleThresh = 0.6;
	icp.distThresh = 0.1;
	icp.icpW = 0.9;
	icp.ICPOnly = false;
	icp.fx = Frame::fx(pyrnum);
	icp.fy = Frame::fy(pyrnum);
	icp.cx = Frame::cx(pyrnum);
	icp.cy = Frame::cy(pyrnum);

	ICPReduceSum_device<<<96, 224>>>(icp);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());

	ReduceSum<float, 29><<<1, MaxThread>>>(sum, result, 96);

	SafeCall(hipDeviceSynchronize());
	SafeCall(hipGetLastError());

	float host_data[29];
	result.download(host_data);
	CreateMatrix(host_data, host_a, host_b);
	cost = sqrt(host_data[27]) / host_data[28];
}
